#include "hip/hip_runtime.h"
#include "util.cuh"

int rowlen(char *riga) {
    int lungh;
    char c;

    lungh = strlen(riga);
    while (lungh > 0) {
        lungh--;
        c = *(riga + lungh);
        if (c == '\0')
            continue;
        if (c == '\40')
            continue; /*  carattere spazio  */
        if (c == '\b')
            continue;
        if (c == '\f')
            continue;
        if (c == '\r')
            continue;
        if (c == '\v')
            continue;
        if (c == '\n')
            continue;
        if (c == '\t')
            continue;
        return (lungh + 1);
    }
    return (0);
}

int readrow(char *rg, int nc, FILE *daleg) {
    int lrg;

    if (fgets(rg, nc, daleg) == NULL)
        return (0);
    lrg = rowlen(rg);
    if (lrg < nc) {
        rg[lrg] = '\0';
        lrg++;
    }
    return (lrg);
}

int MinIntVal(int s, int *a) // OPP: can be ACCELERATEDerated?
{
    int v;
    int e;

    v = a[0];

    for (e = 0; e < s; e++) {
        if (v > a[e])
            v = a[e];
    }

    return v;
}

int MaxIntVal(int s, int *a) // OPP: can be Accelerated?
{
    int v;
    int e;

    int t = 20;

    v = a[0];

    for (e = 0; e < s; e++) {
        if (v < a[e])
            v = a[e];
    }

    return v;
}

double MinDoubleVal(int s, double *a) // OPP: can be Accelerated?
{
    double v;
    int e;

    v = a[0];

    for (e = 0; e < s; e++) {
        if (v > a[e])
            v = a[e];
    }

    return v;
}

double MaxDoubleVal(int s, double *a) // OPP: can be Accelerated?
{
    double v;
    int e;

    v = a[0];

    for (e = 0; e < s; e++) {
        if (v < a[e])
            v = a[e];
    }

    return v;
}

void SetIntValue(int *a, int l, int v) {
    int i;
    for (i = 0; i < l; i++)
        a[i] = v;
    return;
}

void SetDoubleValue(double *a, int l, double v) {
    int i;
    for (i = 0; i < l; i++)
        a[i] = v;
    return;
}

void RealData2ppm(int s1, int s2, double *rdata, double *vmin, double *vmax, char *name) {
    /* Simple subroutine to dump integer data in a PPM format */

    int cm[3][256]; /* R,G,B, Colour Map */
    FILE *ouni, *ColMap;
    int i, j, rc, vp, vs;
    double rmin, rmax;
    char fname[80], jname[80], command[80];

    /* Load color map: 256 colours */
    ColMap = fopen("ColorMap.txt", "r");
    if (ColMap == NULL) {
        fprintf(stderr, "(Error@RealData2ppm) >> Cannot open ColorMap.txt\n");
        exit(-1);
    }
    for (i = 0; i < 256; i++) {
        if (fscanf(ColMap, " %3d %3d %3d", &cm[0][i], &cm[1][i], &cm[2][i]) < 3) {
            fprintf(stderr, "(Error@RealData2ppm) >> reading colour map at line %d: r, g, b =", (i + 1));
            fprintf(stderr, " %3.3d %3.3d %3.3d\n", cm[0][i], cm[1][i], cm[2][i]);
            exit(1);
        }
    }
    fclose(ColMap);

    /* Write on unit 700 with PPM format */
    strcpy(fname, name);
    strcat(fname, ".ppm\0");

    ouni = fopen(fname, "w");
    if (!ouni)
        fprintf(stderr, "(Error@RealData2ppm) >> write access to file %s\n", fname);

    /*  Magic code */
    fprintf(ouni, "P3\n");

    /*  Dimensions */
    fprintf(ouni, "%d %d\n", s1, s2);

    /*  Maximum value */
    fprintf(ouni, "255\n");

    /*  Values from 0 to 255 */
    rmin = MinDoubleVal(s1 * s2, rdata);
    rmax = MaxDoubleVal(s1 * s2, rdata);

    if ((*vmin == *vmax) && (*vmin == (double)0.0)) {
        *vmin = rmin;
        *vmax = rmax;
    } else {
        rmin = *vmin;
        rmax = *vmax;
    }

    vs = 0;
    for (i = 0; i < s1; i++) {
        for (j = 0; j < s2; j++) {
            vp = (int)((rdata[i + (j * s1)] - rmin) * 255.0 / (rmax - rmin));

            if (vp < 0)
                vp = 0;
            if (vp > 255)
                vp = 255;

            vs++;

            fprintf(ouni, " %3.3d %3.3d %3.3d", cm[0][vp], cm[1][vp],
                    cm[2][vp]);

            if (vs >= 10) {
                fprintf(ouni, " \n");
                vs = 0;
            }
        }
        fprintf(ouni, " ");
        vs = 0;
    }
    fclose(ouni);

    return;
}

void Statistics(int s1, int s2, double *rdata, int step) {
    double mnv, mv, mxv, sd;
    int i, j;

    // OPP: Can mean value and standard deviation be computed together?

    // Compute MEAN VALUE
    mv = 0.0;
    mnv = mxv = rdata[0];
    for (i = 0; i < s1; i++) {
        for (j = 0; j < s2; j++) {
            mv = mv + rdata[i + (j * s1)];
            if (mnv > rdata[i + (j * s1)])
                mnv = rdata[i + (j * s1)];
            if (mxv < rdata[i + (j * s1)])
                mxv = rdata[i + (j * s1)];
        }
    }
    mv = mv / (double)(s1 * s2);

    // Compute STANDARD DEVIATION
    sd = 0.0;
    for (i = 0; i < s1; i++) {
        for (j = 0; j < s2; j++) {
            sd = sd + (rdata[i + (j * s1)] - mv) * (rdata[i + (j * s1)] - mv);
        }
    }
    sd = sqrt(sd / (double)(s1 * s2));

    fprintf(
        stdout,
        ">> Step %4d: min, mean, max, std = %12.3e, %12.3e, %12.3e, %12.3e\n",
        step, mnv, mv, mxv, sd);

    return;
}

/* ACCELERATED UTIL */

/* TODO */
