#include "hip/hip_runtime.h"
#include "parallel.cuh"

// Global GPU variables
__device__ int maxRow, maxCol;
__device__ double temp;
__device__ double globalV;
__device__ int iMin;
__device__ int iMax;
__device__ double rMax;
__device__ double rMin;
__device__ double rMean;
__device__ double rStd;

double Sreal, Simag, Rreal, Rimag; // Parameters to compute point sensitiveness -
                                   // values read from input file
int MaxIters;
int TimeSteps;          // Evolution time steps
double *MeasuredValues; // 2-D array - (NumInputValues,3) - Values read in
                        // input file
int NumInputValues;     // Number of values read in input file
double *TheorSlope;     // 2-D array - Theoretical value distribution
int TSlopeLength;       // TheorSlope grid dimensions
int *FieldWeight;       // 2-D array - (Xdots,Ydots) - Degree of sensitiveness to
                        // perturbing field
double *FieldCoord;     // 3-D array - X, Y coordinates in field
double *FieldValues;    // 3-D array - X, Y coordinates in field

void gpuInitGrid(char *InputFile) {
    /* Output:
    !  MeasuredValues(:,3) - values read from input file
    !  Initialization of FieldWeight(Xdots,Ydots) and FieldCoord(Xdots,Ydots,2)
    */

    int valrows, st;
    char filerow[80];
    FILE *inpunit;

    hipError_t err;

    fprintf(stdout, "(NO CPU) >> Initializing grid ...\n");

    inpunit = fopen(InputFile, "r");
    if (!inpunit) {
        fprintf(stderr, "(Error) >>> Cannot access file %s\n", InputFile);
        exit(-1);
    }

    // Read measured values
    NumInputValues = 0;
    valrows = 0;
    while (1) {
        st = readrow(filerow, 80, inpunit);
        if (filerow[0] == '#')
            continue;
        if (NumInputValues <= 0) {
            if (sscanf(filerow, "  %d", &NumInputValues) < 1) {
                if (NumInputValues <= 0) {
                    fprintf(stderr, "(Error) >> there seems to be %d input values...\n", NumInputValues);
                    exit(-1);
                }
            } else {
                MeasuredValues = (double *)malloc(sizeof(double) * NumInputValues * 3);
                if (MeasuredValues == NULL) {
                    fprintf(stderr, "(Error) >> Cannot allocate tmpMeasuredValues[%d,3] :(\n", NumInputValues);
                    exit(-1);
                }
            }
        } else {
            if (sscanf(filerow, "%lf %lf %lf",
                       &MeasuredValues[index2D(valrows, 0, NumInputValues)], // X coord
                       &MeasuredValues[index2D(valrows, 1, NumInputValues)], // Y coord
                       &MeasuredValues[index2D(valrows, 2, NumInputValues)]) // Measured value
                < 3) {
                fprintf(stderr, "(Error) >>> something went wrong while reading MeasuredValues(%d,*)", valrows);
                exit(-1);
            }
            valrows++;
            if (valrows >= NumInputValues)
                break;
        }
    }

    /* Create and initialize FieldWeight */
    err = hipMalloc(&FieldWeight, sizeof(int) * Xdots * Ydots);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error) >> Cannot allocate FieldWeight[%d,%d] on GPU\n", Xdots, Ydots);
        exit(-1);
    }
    hipMemset(FieldWeight, 0, sizeof(int) * Xdots * Ydots);

    /* Create and initialize FieldCoord */
    err = hipMalloc(&FieldCoord, sizeof(double) * Xdots * Ydots * 2);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error) >> Cannot allocate FieldCoord[%d,%d,2] on GPU\n", Xdots, Ydots);
        exit(-1);
    }
    hipMemset(FieldCoord, 0, sizeof(double) * Xdots * Ydots * 2);

    /* Now read Sreal, Simag, Rreal, Rimag */
    Sreal = Simag = Rreal = Rimag = 0.0;
    while (1) {
        if (readrow(filerow, 80, inpunit) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Sreal from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#')
            continue;
        if (sscanf(filerow, "%lf", &Sreal) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Sreal from string.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Simag) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Simag from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rreal) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Rreal from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rimag) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Rimag from input file.\n");
            exit(-1);
        }
        break;
    }

    /* Now read MaxIters */
    MaxIters = 0;
    while (1) {
        if (readrow(filerow, 80, inpunit) < 1) {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1)
            continue;
        if (sscanf(filerow, "%d", &MaxIters) < 1) {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from string.\n");
            exit(-1);
        }
        break;
    }

    /* Now read TimeSteps */
    TimeSteps = 0;
    while (1) {
        if (readrow(filerow, 80, inpunit) < 1) {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1)
            continue;
        if (sscanf(filerow, "%d", &TimeSteps) < 1) {
            fprintf(stderr, "(Error) >> Cannot read TimeSteps from string.\n");
            exit(-1);
        }
        break;
    }

    fclose(inpunit);
    return;
}

__global__ void gpuGridDefKernel(double x0, double y0, double dx, double dy, double *Pts, int Nm1, int len, int TSlopeLength) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    int np, i, j;
    double x, y;

    for (np = id; np < len; np += gridSize) {
        i = np / Nm1;
        j = np % Nm1;

        x = x0 + dx * (double)(i + 1);
        y = y0 + dy * (double)(j + 1);
        Pts[index2D(i, 0, TSlopeLength)] = x;
        Pts[index2D(i, 1, TSlopeLength)] = y;
    }
}

void gpuGridDef(double x0, double x1, double y0, double y1, int N, double *Pts) {
    double dx, dy;
    int Mm1, Nm1;

    Mm1 = sqrt((double)Xdots) - 1;
    Nm1 = sqrt((double)Ydots) - 1;
    dx = (x1 - x0) / (double)N;
    dy = (y1 - y0) / (double)N;

    gpuGridDefKernel<<<6, 128>>>(x0, y0, dx, dy, Pts, Nm1, Nm1 * Mm1, TSlopeLength);

    return;
}

__device__ double gpuSolution(double x, double y) {
    return ((x * x * x) + (y * y * y)) / (double)6.0;
}

__global__ void gpuEqsDefKernel(double x0, double x1, double y0, double y1, int Nm1, double dx, double dy, int LA, double *A, double *Rhs, double *Pts, int TSlopeLength) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    int np, pos;
    double x, y, Eps;

    for (np = id; np < LA; np += gridSize) {
        x = Pts[index2D(np, 0, TSlopeLength)];
        y = Pts[index2D(np, 1, TSlopeLength)];

        A[index2D(np, np, LA)] = -4.0;

        Rhs[np] = (x + y) * dx * dy;

        // define Eps function of grid dimensions
        Eps = (dx + dy) / 20.0;

        // where is P(x-dx,y) ?
        if (fabs((x - dx) - x0) < Eps)
            Rhs[np] = Rhs[np] - gpuSolution(x0, y);
        else {
            // Find pos = position of P(x-dx,y)
            pos = np - Nm1;
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x+dx,y) ?
        if (fabs((x + dx) - x1) < Eps)
            Rhs[np] = Rhs[np] - gpuSolution(x1, y);
        else {
            // Find pos = position of P(x+dx,y)
            pos = np + Nm1;
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y-dy) ?
        if (fabs((y - dy) - y0) < Eps)
            Rhs[np] = Rhs[np] - gpuSolution(x, y0);
        else {
            // Find pos = position of P(x,y-dy)
            pos = np - 1;
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y+dy) ?
        if (fabs((y + dy) - y1) < Eps)
            Rhs[np] = Rhs[np] - gpuSolution(x, y1);
        else {
            // Find pos = position of P(x,y-dy)
            pos = np + 1;
            A[index2D(np, pos, LA)] = 1.0;
        }
    }
}

void gpuEqsDef(double x0, double x1, double y0, double y1, int N, int LA, double *A, double *Rhs, double *Pts) {
    // Pts(LA,3) - inner grid point Coordinates
    // Rhs(LA)   - Linear equation Right Hand Side
    // A(LA,LA)  - Linear equation matrix

    double x, y, Eps, dx, dy;
    int np, Nm1, pos;

    //  Define A matrix and RHS

    Nm1 = N - 1;
    dx = (x1 - x0) / (double)N;
    dy = (y1 - y0) / (double)N;

    hipMemset(A, 0, sizeof(double) * LA * LA);
    hipMemset(Rhs, 0, sizeof(double) * LA);

    gpuEqsDefKernel<<<6, 128>>>(x0, x1, y0, y1, Nm1, dx, dy, LA, A, Rhs, Pts, TSlopeLength);

    return;
}

/*
 * result in d_b. d_A contains L matrix of LU factorization
 */
int LinEquSolve_ACC(double *d_A, // dense coefficient matrix (on device)
                    int n,       // size (square)
                    double *d_b) // A*x = b  (on device)
{
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;
    int rowsA = n;      // number of rows of A
    int colsA = n;      // number of columns of A
    int lda = n;        // leading dimension in dense matrix
    double *h_r = NULL; // r = b - A*x, copy of d_r

    // double *d_x = NULL; // x = A \ h_b, saved in d_b
    // double *d_r = NULL; // r = b - A*x

    // cuSolver setup
    checkCudaErrors(hipsolverDnCreate(&handle));
    checkCudaErrors(hipStreamCreate(&stream));

    // hipblasHandle_t cublasHandle = NULL; // used in residual evaluation
    // checkCudaErrors(hipsolverSetStream(handle, stream));
    // checkCudaErrors(hipblasSetStream(cublasHandle, stream));

    // allocate on device
    // checkCudaErrors(hipMalloc((void **)&d_r, sizeof(double) * rowsA));

    // actually solve
    linearSolverLU(handle, rowsA, d_A, lda, d_b);

    checkCudaErrors(hipDeviceSynchronize());

    return 0;
}

double *gpuFieldDistribution() {
    /*
    !  Compute theoretical value distribution of the perturbing field
    !  Output: TheorSlope(TSlopeLength,3) - theoretical field distribution function
    */
    double *CoeffMatrix, *B;
    double x0, y0, x1, y1;
    double t0, t1;

    int M, Mm1, N, Nm1, LA;
    int i, rc;

    hipError_t err;

    fprintf(stdout, "\t>> Computing theoretical perturbing field...\n");

    x0 = Sreal;
    y0 = Simag;
    x1 = x0 + Rreal;
    y1 = y0 + Rimag;

    // How many intervals? It should be safe to use SQRT(Xdots)
    M = sqrt((double)Xdots);
    N = sqrt((double)Ydots);

    Nm1 = N - 1; // Grid points minus boundary
    Mm1 = M - 1; // Grid points minus boundary

    LA = Mm1 * Nm1; // unknown points
    TSlopeLength = LA;

    /* Allocate CoeffMatrix */
    err = hipMalloc(&CoeffMatrix, sizeof(double) * LA * LA);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error) >> Cannot allocate CoeffMatrix[%d,%d] on GPU\n", LA, LA);
        exit(-1);
    }

    /* Allocate TheorSlope */
    err = hipMalloc(&TheorSlope, sizeof(double) * TSlopeLength * 3);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error) >> Cannot allocate TheorSlope[%d,3] on GPU\n", TSlopeLength);
        exit(-1);
    }

    /* Allocate B */
    err = hipMalloc(&B, sizeof(double) * LA);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error) >> Cannot allocate B[%d] on GPU\n", LA);
        exit(-1);
    }

    gpuGridDef(x0, x1, y0, y1, N, TheorSlope);
    hipDeviceSynchronize();

    gpuEqsDef(x0, x1, y0, y1, N, LA, CoeffMatrix, B, TheorSlope);
    hipDeviceSynchronize();

    // gpuLinEquSolve(CoeffMatrix, LA, B);
    t0 = second();
    rc = LinEquSolve_ACC(CoeffMatrix, LA, B);
    // rc = gpuLinEquSolve(CoeffMatrix, LA, B);
    t1 = second();
    fprintf(stdout, "\t>> LinEquSolve took %lf seconds\n", (t1 - t0));

    if (rc != 0)
        exit(-1); // TODO
    hipDeviceSynchronize();

    hipMemcpy(&TheorSlope[2 * TSlopeLength], B, sizeof(double) * LA, hipMemcpyDeviceToDevice);

    hipFree(CoeffMatrix);
    double *CPU_B = (double *)malloc(sizeof(double) * LA);
    hipMemcpy(CPU_B, B, sizeof(double) * LA, hipMemcpyDeviceToHost);
    hipFree(B);
    hipDeviceSynchronize();

    return CPU_B;
}

__global__ void gpuSensiblePointsKernel(double Ir, double Ii, double Xinc, double Yinc, int MaxIt, double *FieldCoord, int *FieldWeight) {

    double ca, cb, za, zb;
    double rad, zan, zbn;

    int ix, iy, iz;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    int sizeX = gridDim.x * blockDim.x;
    int sizeY = gridDim.y * blockDim.y;

    for (iy = idy; iy < Ydots; iy += sizeY) {
        for (ix = idx; ix < Xdots; ix += sizeX) {

            ca = Xinc * ix + Ir;
            cb = Yinc * iy + Ii;
            FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)] = ca;
            FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)] = cb;

            rad = ca * ca + cb * cb;

            zan = 0.0;
            zbn = 0.0;

            for (iz = 1; iz <= MaxIt; iz++) {

                if (rad > (double)4.0)
                    break;

                za = zan;
                zb = zbn;

                zan = ca + (za - zb) * (za + zb);
                zbn = 2.0 * za * zb + cb;

                rad = zan * zan + zbn * zbn;
            }

            FieldWeight[index2D(ix, iy, Xdots)] = iz;
        }
    }
}

void gpuSensiblePoints(double Ir, double Ii, double Sr, double Si, int MaxIt) {
    /*
    !  Compute "heated" points
    !  Output:
    !          FieldCoord(Xdots,Ydots,2)
    !          FieldWeight(Xdots,Ydots)
   */

    double Xinc, Yinc;

    hipError_t err;

    fprintf(stdout, "\t>> Computing sensitivity to field effects...\n");

    Xinc = Sr / (double)Xdots;
    Yinc = Si / (double)Ydots;

    gpuSensiblePointsKernel<<<dim3(8, 8, 1), dim3(8, 8, 1)>>>(Ir, Ii, Xinc, Yinc, MaxIt, FieldCoord, FieldWeight);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
        printf("(CUDA Error) >> %s\n", hipGetErrorString(err));

    return;
}

void gpuFieldInit() {
    /*
    ! Initialize field values in the grid. Values are computed on the basis
    ! of the measured values read in subroutine InitGrid and the gross grid
    ! values computed in subroutine FieldDistribution. Moreover sensitiveness
    ! to field effects as computed in subroutine SensiblePoints are taken into
    ! account.
    !
    ! Input:
    !        MeasuredValues(:,3)
    !        FieldWeight(Xdots,Ydots)
    ! Output:
    !        FieldValues(Xdots,Ydots,2)
    */

    int rv;
    double xc, yc, ev, sv, sd, DiscrValue;
    double *DiffValues;

    hipError_t err;

    fprintf(stdout, "\t>> Initializing entity of field effects...\n");

    /* Allocate FieldValues */
    err = hipMalloc(&FieldValues, sizeof(double) * Xdots * Ydots * 2);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate FieldValues[%d,%d,2] in GPU\n", Xdots, Ydots);
        exit(-1);
    }
    hipMemset(FieldValues, 0, sizeof(double) * Xdots * Ydots * 2);

    /* Allocate DiffValues */
    DiffValues = (double *)malloc(sizeof(double) * NumInputValues);
    if (DiffValues == NULL) {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate DiffValues[%d]\n", NumInputValues);
        exit(-1);
    }
    memset(DiffValues, 0, sizeof(double) * NumInputValues);

    /* Compute discrepancy between Measured and Theoretical value */

    DiscrValue = 0.0;
    for (rv = 0; rv < NumInputValues; rv++) {
        xc = MeasuredValues[index2D(rv, 0, NumInputValues)];
        yc = MeasuredValues[index2D(rv, 1, NumInputValues)];

        // TheorSlope is computed on the basis of a coarser grid, so look for the best values near xc, yc coordinates
        sv = gpuNearestValue(xc, yc, TSlopeLength, TheorSlope);
        ev = MeasuredValues[index2D(rv, 2, NumInputValues)];

        DiffValues[rv] = ev - sv;
        DiscrValue += ev - sv;
    }
    DiscrValue = DiscrValue / (double)NumInputValues;

    // Compute standard deviation
    sd = 0.0;
    for (rv = 0; rv < NumInputValues; rv++)
        sd = sd + (DiffValues[rv] - DiscrValue) * (DiffValues[rv] - DiscrValue);
    sd = sqrt(sd / (double)NumInputValues);

    // Print statistics
    fprintf(stdout, "\t...Number of Points, Mean value, Standard deviation = %d, %12.3e, %12.3e\n", NumInputValues, DiscrValue, sd);

    // Compute FieldValues stage 1

    gpuFieldPoints(DiscrValue);

    free(DiffValues);
}

void gpuCooling(int steps) {
    /*
    !  Compute evolution of the effects of the field
    !  Input/Output:
    !                FieldValues(Xdots,Ydots,2)
    */

    int iz, it;
    char fname[80];
    double vmin, vmax;

    double *tmp;
    int reduceLayer = (Xdots * Ydots + 1) / 2;

    hipError_t err;

    /* Allocate space for temporary results */
    err = hipMalloc(&tmp, sizeof(double) * reduceLayer * 4);
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> %s\n", hipGetErrorString(err));
        return;
    }

    // --------------

    fprintf(stdout, "\t>> Computing cooling of field effects ...\n");
    fprintf(stdout, "\t... %d steps ...\n", steps);
    sprintf(fname, "FieldValues0000");

    vmin = vmax = 0.0;
    // RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], &vmin, &vmax, fname);
    gpuStatistics(Xdots, Ydots, FieldValues, tmp, 0);

    iz = 1;
    for (it = 1; it <= steps; it++) {
        // Update the value of grid points
        gpuUpdate(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &FieldValues[index3D(0, 0, 2 - iz, Xdots, Ydots)]);
        hipDeviceSynchronize();

        iz = 3 - iz;

        // Print and show results
        sprintf(fname, "FieldValues%4.4d", it);
        // if (it % 4 == 0) RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &vmin, &vmax, fname);
        gpuStatistics(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], tmp, it);
    }

    hipFree(tmp);

    return;
}

int gpuLinEquSolve(double *a, int n, double *b) {
    /* Gauss-Jordan elimination algorithm */
    int *indcol, *indrow, *ipiv;

    hipError_t err;

    /* Allocate indcol */
    err = hipMalloc(&indcol, sizeof(int) * n);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indcol[%d] on GPU\n", n);
        return (-1);
    }

    /* Allocate indrow */
    err = hipMalloc(&indrow, sizeof(int) * n);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indrow[%d] on GPU\n", n);
        return (-1);
    }

    /* Allocate ipiv */
    err = hipMalloc(&ipiv, sizeof(int) * n);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate ipiv[%d] on GPU\n", n);
        return (-1);
    }
    hipMemset(ipiv, 0, sizeof(int) * n);

    /* Actual algorithm */

    int *maxIndex;
    double *maxima;

    err = hipMalloc(&maxIndex, sizeof(int) * n);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate maxIndex on GPU\n", n);
        return (-1);
    }

    err = hipMalloc(&maxima, sizeof(double) * n);
    if (err != hipSuccess) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate maxima on GPU\n", n);
        return (-1);
    }

    gpuLinEquSolveKernel<<<32, 256>>>(maxima, maxIndex, a, b, indrow, indcol, ipiv, n);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
        printf("(CUDA Error) >> %s\n", hipGetErrorString(err));

    hipFree(indcol);
    hipFree(indrow);
    hipFree(ipiv);
    hipFree(maxIndex);
    hipFree(maxima);

    return 0;
}

__global__ void gpuLinEquSolveKernel(double *maxima, int *maxIndex, double *a, double *b, int *indrow, int *indcol, int *ipiv, int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    int iter, i, j, k, icol;
    double max, tmp;

    for (iter = 0; iter < n; iter++) {
        for (i = id; i < n; i += gridSize) {
            max = 0;
            icol = 0;

            if (ipiv[i] != 1) {
                for (j = 0; j < n; j++) {
                    if (ipiv[j] == 0 && max <= fabs(a[index2D(i, j, n)])) {
                        max = fabs(a[index2D(i, j, n)]);
                        icol = j;
                    }
                }
            }

            maxima[i] = max;
            maxIndex[i] = icol;
        }

        __syncthreads();

        if (id == 0) {
            j = 0;

            for (i = 1; i < n; i++)
                if (maxima[i] > maxima[j])
                    j = i;

            maxRow = j;
            maxCol = maxIndex[j];

            ipiv[maxCol] = ipiv[maxCol] + 1;
        }

        __syncthreads();

        if (maxRow != maxCol) {
            for (i = id; i < n; i += gridSize) {
                tmp = a[index2D(maxRow, i, n)];
                a[index2D(maxRow, i, n)] = a[index2D(maxCol, i, n)];
                a[index2D(maxCol, i, n)] = tmp;
            }

            if (id == 0) {
                tmp = b[maxRow];
                b[maxRow] = b[maxCol];
                b[maxCol] = tmp;
            }
        }

        if (id == 0) {
            indrow[iter] = maxRow;
            indcol[iter] = maxCol;
        }

        __syncthreads();

        // TODO: Missing check on singularity

        if (id == 0) {
            temp = a[index2D(maxCol, maxCol, n)];
            a[index2D(maxCol, maxCol, n)] = 1.0;
            b[maxCol] /= temp;
        }

        for (i = id; i < n; i += gridSize) {
            a[index2D(maxCol, i, n)] /= temp;
        }

        __syncthreads();

        for (i = id; i < n; i += gridSize) {
            if (i != maxCol) {
                tmp = a[index2D(i, maxCol, n)];
                a[index2D(i, maxCol, n)] = 0.0;
                for (k = 0; k < n; k++) {
                    a[index2D(i, k, n)] = a[index2D(i, k, n)] - a[index2D(maxCol, k, n)] * tmp;
                }
                b[i] = b[i] - b[maxCol] * tmp;
            }
        }

        __syncthreads();
    }
}

double gpuNearestValue(double xc, double yc, int ld, double *Values) {
    double v;

    double *dist;
    int *mask;
    double *partialResult;

    hipError_t err;

    /* Allocate dist */
    err = hipMalloc(&dist, sizeof(double) * ld);
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> Cannot allocate dist in GPU\n");
        exit(-1);
    }

    /* Allocate mask */
    err = hipMalloc(&mask, sizeof(int) * ld);
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> Cannot allocate mask in GPU\n");
        exit(-1);
    }

    /* Allocate partialResult */
    err = hipMalloc(&partialResult, sizeof(double) * ld);
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> Cannot allocate partialResult in GPU\n");
        exit(-1);
    }

    gpuNearestValueKernel<<<1, 768>>>(xc, yc, Values, dist, mask, partialResult, ld);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> %s\n", hipGetErrorString(err));
        exit(-1);
    }

    hipMemcpyFromSymbol(&v, HIP_SYMBOL(globalV), sizeof(double), 0, hipMemcpyDeviceToHost);

    hipFree(dist);
    hipFree(mask);
    hipFree(partialResult);

    return v;
}

__global__ void gpuNearestValueKernel(double xc, double yc, double *Values, double *dist, int *mask, double *partialResult, int n) {

    int i, j;
    double a, b, v;
    __shared__ int np;
    __shared__ double md;

    // Compute distances

    for (i = threadIdx.x; i < n; i += blockDim.x) {
        a = xc - Values[index2D(i, 0, n)];
        b = yc - Values[index2D(i, 1, n)];
        dist[i] = a * a + b * b;
    }
    __syncthreads();

    // Compute Minimum

    int last;
    int reduceLayer = (n + 1) / 2;

    for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {

        if ((2 * i + 1) < n) {

            a = dist[2 * i];
            b = dist[2 * i + 1];

            if (a < b) {
                // dist[2 * i] = a;
                mask[2 * i] = 1;
                partialResult[2 * i] = Values[index2D(2 * i, 2, n)];
            } else if (b < a) {
                dist[2 * i] = b;
                mask[2 * i] = 1;
                partialResult[2 * i] = Values[index2D(2 * i + 1, 2, n)];
            } else {
                // dist[2 * i] = a;
                mask[2 * i] = 2;
                partialResult[2 * i] = Values[index2D(2 * i, 2, n)] + Values[index2D(2 * i + 1, 2, n)];
            }
        } else {
            mask[2 * i] = 1;
            partialResult[2 * i] = Values[index2D(2 * i, 2, n)];
        }
    }

    __syncthreads();

    for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {
        dist[i] = dist[2 * i];
        mask[i] = mask[2 * i];
        partialResult[i] = partialResult[2 * i];
    }

    last = reduceLayer % 2;
    reduceLayer = (reduceLayer + 1) / 2;

    __syncthreads();

    // Reducing Part

    while (reduceLayer > 1) {
        for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {
            if (i < reduceLayer - last) {

                a = dist[2 * i];
                b = dist[2 * i + 1];

                if (b < a) {
                    dist[2 * i] = b;
                    mask[2 * i] = mask[2 * i + 1];
                    partialResult[2 * i] = partialResult[2 * i + 1];
                } else if (a == b) {
                    // dist[2 * i] = a;
                    mask[2 * i] += mask[2 * i + 1];
                    partialResult[2 * i] += partialResult[2 * i + 1];
                }
            }
        }

        __syncthreads();

        for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {
            dist[i] = dist[2 * i];
            mask[i] = mask[2 * i];
            partialResult[i] = partialResult[2 * i];
        }

        last = reduceLayer % 2;
        reduceLayer = (reduceLayer + 1) / 2;

        __syncthreads();
    }

    // Compute final result

    if (threadIdx.x == 0) {

        a = dist[0];
        b = dist[1];

        if (a == b)
            globalV = (partialResult[0] + partialResult[1]) / (double)(mask[0] + mask[1]);
        else if (a < b)
            globalV = partialResult[0] / (double)mask[0];
        else if (b < a)
            globalV = partialResult[1] / (double)mask[1];
    }
}

void gpuFieldPoints(double Diff) {
    hipError_t err;

    MinMaxIntVal(FieldWeight, Xdots * Ydots);

    gpuFieldPointsKernel<<<dim3(16, 16, 1), dim3(32, 32, 1)>>>(FieldCoord, FieldWeight, FieldValues, Diff, TSlopeLength, TheorSlope);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> %s\n", hipGetErrorString(err));
    }

    hipMemcpy(&FieldValues[Xdots * Ydots], FieldValues, sizeof(double) * Xdots * Ydots, hipMemcpyDeviceToDevice);

    return;
}

void MinMaxIntVal(int *Values, int len) {
    int *tmpMax;
    int *tmpMin;

    hipError_t err;

    /* Allocate Temporary Results */
    err = hipMalloc(&tmpMax, sizeof(int) * (len + 1) / 2);
    if (err != hipSuccess) {
        fprintf(stderr, "(hipError_t) >>> %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMalloc(&tmpMin, sizeof(int) * (len + 1) / 2);
    if (err != hipSuccess) {
        fprintf(stderr, "(hipError_t) >>> %s\n", hipGetErrorString(err));
        return;
    }

    MinMaxIntValKernel<<<1, 768>>>(Values, len, tmpMin, tmpMax);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "(hipError_t) >>> %s\n", hipGetErrorString(err));
        return;
    }

    hipFree(tmpMax);
    hipFree(tmpMin);
}

__global__ void MinMaxIntValKernel(int *Values, int len, int *tmpMin, int *tmpMax) {

    int i;
    double a, b;

    // Compute

    int last;
    int reduceLayer = (len + 1) / 2;

    for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {

        a = Values[2 * i];

        if ((2 * i + 1) < len) {

            b = Values[2 * i + 1];

            if (a <= b) {
                tmpMin[i] = a;
                tmpMax[i] = b;
            } else {
                tmpMin[i] = b;
                tmpMax[i] = a;
            }
        } else {
            tmpMin[i] = a;
            tmpMax[i] = a;
        }
    }

    __syncthreads();

    last = reduceLayer % 2;
    reduceLayer = (reduceLayer + 1) / 2;

    __syncthreads();

    // Reducing Part

    while (reduceLayer > 1) {
        for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {
            if (i < reduceLayer - last) {

                a = tmpMin[2 * i];
                b = tmpMin[2 * i + 1];

                if (b < a)
                    tmpMin[2 * i] = b;

                a = tmpMax[2 * i];
                b = tmpMax[2 * i + 1];

                if (b > a)
                    tmpMax[2 * i] = b;
            }
        }

        __syncthreads();

        for (i = threadIdx.x; i < reduceLayer; i += blockDim.x) {
            tmpMin[i] = tmpMin[2 * i];
            tmpMax[i] = tmpMax[2 * i];
        }

        last = reduceLayer % 2;
        reduceLayer = (reduceLayer + 1) / 2;

        __syncthreads();
    }

    // Compute final result

    if (threadIdx.x == 0) {

        a = tmpMin[0];
        b = tmpMin[1];

        iMin = (a <= b) ? a : b;

        a = tmpMax[0];
        b = tmpMax[1];

        iMax = (a >= b) ? a : b;

        printf("-----> iMin = %d, iMax = %d on GPU\n", iMin, iMax);
    }

    /*int i;
    double a, b;
    int layerLength = (len + 1) / 2;
    int last;

    for (i = threadIdx.x; i < layerLength; i += blockDim.x) {
        if ((2 * i + 1) < len) {
            a = Values[2 * i];
            b = Values[2 * i + 1];

            tmpMin[i] = (a < b) ? a : b;
            tmpMax[i] = (a > b) ? a : b;
        } else {
            tmpMin[i] = Values[2 * i];
            tmpMax[i] = Values[2 * i];
        }
    }

    last = layerLength % 2;
    layerLength = (layerLength + 1) / 2;

    __syncthreads();

    while (layerLength > 1) {
        for (i = threadIdx.x; i < layerLength; i += blockDim.x) {
            if (i < layerLength - last) {
                a = tmpMin[2 * i];
                b = tmpMin[2 * i + 1];
                tmpMin[2 * i] = (a < b) ? a : b;

                a = tmpMax[2 * i];
                b = tmpMax[2 * i + 1];
                tmpMax[2 * i] = (a > b) ? a : b;
            }
        }

        __syncthreads();

        for (i = threadIdx.x; i < layerLength; i += blockDim.x) {
            tmpMin[i] = tmpMin[2 * i];
            tmpMax[i] = tmpMax[2 * i];
        }

        last = layerLength % 2;
        layerLength = (layerLength + 1) / 2;

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        iMin = tmpMin[0];
        iMax = tmpMax[0];


        printf("-----> iMin = %d, iMax = %d on GPU\n", iMin, iMax);
    }*/
}

__global__ void gpuFieldPointsKernel(double *FieldCoord, int *FieldWeight, double *FieldValues, double Diff, int TSlopeLength, double *TheorSlope) {
    int iy, ix;
    double xc, yc, sv;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int sizeX = blockDim.x * gridDim.x;
    int sizeY = blockDim.y * gridDim.y;

    for (iy = idy; iy < Ydots; iy += sizeY) {
        for (ix = idx; ix < Xdots; ix += sizeX) {
            xc = FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)];
            yc = FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)];

            // Compute effects of field in every point
            sv = deviceNearestValue(xc, yc, TSlopeLength, TheorSlope);
            FieldValues[index3D(ix, iy, 0, Xdots, Ydots)] = 293.16 + 80 * (Diff + sv) * (FieldWeight[index2D(ix, iy, Xdots)] - iMin) / (iMax - iMin);
        }
    }
}

__device__ double deviceNearestValue(double xc, double yc, int ld, double *Values) {

    // look for the best values near xc, yc coordinates
    double v;

    double d, md; // minimum distance
    int np;       // number of nearest points
    int i;

    md = ((xc - Values[index2D(0, 0, ld)]) * (xc - Values[index2D(0, 0, ld)])) +
         ((yc - Values[index2D(0, 1, ld)]) * (yc - Values[index2D(0, 1, ld)]));

    np = 1;
    v = Values[index2D(0, 2, ld)];

    // Compute lowest distance
    for (i = 1; i < ld; i++) {

        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));

        if (d == md) {
            np++;
            v += Values[index2D(i, 2, ld)];
        } else if (d < md) {
            md = d;
            np = 1;
            v = Values[index2D(i, 2, ld)];
        }
    }

    // mean value
    v = v / (double)np;

    return v;
}

void gpuUpdate(int xdots, int ydots, double *u1, double *u2) {
    /* Compute next step using matrices g1, g2 of dimension (nr,nc) */

    int i, j;
    double CX, CY;
    double hx, dgx, hy, dgy, dd;

    dd = 0.0000001;
    hx = 1.0 / (double)xdots;
    hy = 1.0 / (double)ydots;
    dgx = -2.0 + hx * hx / (2 * dd);
    dgy = -2.0 + hy * hy / (2 * dd);
    CX = dd / (hx * hx);
    CY = dd / (hy * hy);

    gpuUpdateKernel<<<dim3(16, 16, 1), 128>>>(xdots, ydots, u1, u2, CX, CY, dgx, dgy);

    return;
}

__global__ void gpuUpdateKernel(int xdots, int ydots, double *u1, double *u2, double CX, double CY, double dgx, double dgy) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int sizeX = blockDim.x * gridDim.x;
    int sizeY = blockDim.y * gridDim.y;

    int i, j;

    for (j = idy; j < ydots - 1; j += sizeY) {
        for (i = idx; i < xdots - 1; i += sizeX) {
            if (i <= 0 || i >= xdots - 1) {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            if (j <= 0 || j >= ydots - 1) {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            u2[index2D(i, j, xdots)] = CX * (u1[index2D((i - 1), j, xdots)] + u1[index2D((i + 1), j, xdots)] + dgx * u1[index2D((i + 1), j, xdots)]) + CY * (u1[index2D(i, (j - 1), xdots)] + u1[index2D(i, (j + 1), xdots)] + dgy * u1[index2D(i, j, xdots)]);
        }
    }

    __syncthreads();

    for (j = idy; j < ydots - 1; j += sizeY) {
        u2[index2D(0, j, xdots)] = u2[index2D(1, j, xdots)];
        u2[index2D(Xdots - 1, j, xdots)] = u2[index2D(Xdots - 2, j, xdots)];
    }

    for (i = idx; i < xdots - 1; i += sizeX) {
        u2[index2D(i, 0, xdots)] = u2[index2D(i, 1, xdots)];
        u2[index2D(i, Ydots - 1, xdots)] = u2[index2D(i, Ydots - 2, xdots)];
    }
}

void gpuStatistics(int s1, int s2, double *rdata, double *tmp, int step) {

    double mnv, mv, mxv, sd;

    double *tmpMin;
    double *tmpMax;
    double *tmpMean;
    double *tmpStd;

    hipError_t err;

    int reduceLayer = (s1 * s2 + 1) / 2;

    tmpMin = tmp;
    tmpMax = &tmpMin[reduceLayer];
    tmpMean = &tmpMax[reduceLayer];
    tmpStd = &tmpMean[reduceLayer];

    bulkReduce<<<256, 128>>>(tmpMin, tmpMax, tmpMean, rdata, s1 * s2);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> %s\n", hipGetErrorString(err));
        return;
    }

    gpuStatisticsKernel<<<1, 1024>>>(tmpMin, tmpMax, tmpMean, tmpStd, rdata, s1 * s2, 256 * 128);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "(CUDA Error) >> %s\n", hipGetErrorString(err));
        return;
    }

    hipMemcpyFromSymbol(&mnv, HIP_SYMBOL(rMin), sizeof(double), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&mv, HIP_SYMBOL(rMean), sizeof(double), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&mxv, HIP_SYMBOL(rMax), sizeof(double), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sd, HIP_SYMBOL(rStd), sizeof(double), 0, hipMemcpyDeviceToHost);

    fprintf(stdout, ">> Step %4d: min, mean, max, std = %12.3e, %12.3e, %12.3e, %12.3e\n", step, mnv, mv, mxv, sd);

    return;
}

__global__ void bulkReduce(double *tmpMin, double *tmpMax, double *tmpMean, double *Values, int len) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int i;
    double a, min, max, mean;

    // Strong reduce

    int jobShare = len / stride;
    int remainderJob = len % stride;

    int offset = id * jobShare;

    jobShare += (id < remainderJob);
    offset += (id < remainderJob) ? id : remainderJob;

    if (jobShare > 1) {

        min = Values[offset];
        max = Values[offset];
        mean = Values[offset];

        for (i = 1; i < jobShare; i++) {

            if ((offset + i) >= len)
                break;

            a = Values[offset + i];

            if (a < min)
                min = a;
            if (a > max)
                max = a;
            mean += a;
        }

        tmpMin[id] = min;
        tmpMax[id] = max;
        tmpMean[id] = mean;
    }
}

__global__ void gpuStatisticsKernel(double *tmpMin, double *tmpMax, double *tmpMean, double *tmpStd, double *Values, int len, int reduceLayer) {

    int id = threadIdx.x;
    int stride = blockDim.x;

    int i;
    double a, b, min, max, mean;
    int layerLength = reduceLayer;

    int last;

    // Strong reduce

    int jobShare = reduceLayer / blockDim.x;
    int remainderJob = reduceLayer % blockDim.x;

    int offset = id * jobShare;

    jobShare += (id < remainderJob);
    offset += (id < remainderJob) ? id : remainderJob;

    if (jobShare > 1) {

        min = tmpMin[offset];
        max = tmpMax[offset];
        mean = tmpMean[offset];

        for (i = 1; i < jobShare; i++) {

            if ((offset + i) >= reduceLayer)
                break;

            if (tmpMin[offset + i] < min)
                min = tmpMin[offset + i];
            if (tmpMax[offset + i] > max)
                max = tmpMax[offset + i];
            mean += tmpMean[offset + i];
        }

        __syncthreads();

        tmpMin[id] = min;
        tmpMax[id] = max;
        tmpMean[id] = mean;

        layerLength = blockDim.x;
    }

    // Further Reduce

    last = layerLength % 2;
    layerLength = (layerLength + 1) / 2;

    __syncthreads();

    while (layerLength > 1) {
        for (i = id; i < layerLength; i += stride) {
            if (i < layerLength - last) {
                a = tmpMin[2 * i];
                b = tmpMin[2 * i + 1];
                tmpMin[2 * i] = (a < b) ? a : b;

                a = tmpMax[2 * i];
                b = tmpMax[2 * i + 1];
                tmpMax[2 * i] = (a > b) ? a : b;

                a = tmpMean[2 * i];
                b = tmpMean[2 * i + 1];
                tmpMean[2 * i] = a + b;
            }
        }

        __syncthreads();

        for (i = id; i < layerLength; i += stride) {
            tmpMin[i] = tmpMin[2 * i];
            tmpMax[i] = tmpMax[2 * i];
            tmpMean[i] = tmpMean[2 * i];
        }

        last = layerLength % 2;
        layerLength = (layerLength + 1) / 2;

        __syncthreads();
    }

    if (id == 0) {
        a = tmpMin[0];
        b = tmpMin[1];
        rMin = (a < b) ? a : b;

        a = tmpMax[0];
        b = tmpMax[1];
        rMax = (a > b) ? a : b;

        a = tmpMean[0];
        b = tmpMean[1];
        rMean = (a + b) / (double)len;
    }

    __syncthreads();

    // Compute STD

    double mv = rMean;

    layerLength = reduceLayer;

    for (i = id; i < layerLength; i += stride) {
        if ((2 * i + 1) < len) {
            a = Values[2 * i];
            b = Values[2 * i + 1];

            a = (a - mv) * (a - mv);
            b = (b - mv) * (b - mv);

            tmpStd[i] = a + b;
        } else {
            a = Values[2 * i];
            a = (a - mv) * (a - mv);

            tmpStd[i] = a;
        }
    }

    last = layerLength % 2;
    layerLength = (layerLength + 1) / 2;

    __syncthreads();

    while (layerLength > 1) {
        for (i = id; i < layerLength; i += stride) {
            if (i < layerLength - last) {
                a = tmpStd[2 * i];
                b = tmpStd[2 * i + 1];
                tmpStd[2 * i] = a + b;
            }
        }

        __syncthreads();

        for (i = id; i < layerLength; i += stride)
            tmpStd[i] = tmpStd[2 * i];

        last = layerLength % 2;
        layerLength = (layerLength + 1) / 2;

        __syncthreads();
    }

    if (id == 0) {
        rStd = sqrt((tmpStd[0] + tmpStd[1]) / (double)len);
    }
}
