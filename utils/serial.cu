#include "hip/hip_runtime.h"
#include "serial.h"

void InitGrid(char *InputFile) {
    /* Output:
    !  MeasuredValues(:,3) - values read from input file
    !  Initialization of FieldWeight(Xdots,Ydots) and FieldCoord(Xdots,Ydots,2)
    */

    int valrows, st;
    char filerow[80];
    FILE *inpunit;

    fprintf(stdout, ">> Initializing grid ...\n");

    inpunit = fopen(InputFile, "r");
    if (!inpunit) {
        fprintf(stderr, "(Error) >>> Cannot access file %s\n", InputFile);
        exit(-1);
    }

    // Read measured values
    NumInputValues = 0;
    valrows = 0;
    while (1) {
        st = readrow(filerow, 80, inpunit);
        if (filerow[0] == '#')
            continue;
        if (NumInputValues <= 0) {
            if (sscanf(filerow, "  %d", &NumInputValues) < 1) {
                if (NumInputValues <= 0) {
                    fprintf(stderr, "(Error) >> there seems to be %d input values...\n", NumInputValues);
                    exit(-1);
                }
            } else {
                MeasuredValues = (double *)malloc(sizeof(double) * NumInputValues * 3);
                if (MeasuredValues == NULL) {
                    fprintf(stderr, "(Error) >> Cannot allocate MeasuredValues[%d,3] :(\n", NumInputValues);
                    exit(-1);
                }
            }
        } else {
            if (sscanf(filerow, "%lf %lf %lf",
                       &MeasuredValues[index2D(valrows, 0, NumInputValues)], // X coord
                       &MeasuredValues[index2D(valrows, 1, NumInputValues)], // Y coord
                       &MeasuredValues[index2D(valrows, 2, NumInputValues)]) // Measured value
                < 3) {
                fprintf(stderr, "(Error) >>> something went wrong while reading MeasuredValues(%d,*)", valrows);
                exit(-1);
            }
            valrows++;
            if (valrows >= NumInputValues)
                break;
        }
    }

    /* Create and initialize FieldWeight */
    FieldWeight = (int *)malloc(sizeof(int) * Xdots * Ydots);
    if (FieldWeight == NULL) {
        fprintf(stderr, "(Error) >> Cannot allocate FieldWeight[%d,%d]\n", Xdots, Ydots);
        exit(-1);
    }
    SetIntValue(FieldWeight, Xdots * Ydots, 0); // OPP: you can use calloc?

    /* Create and initialize FieldCoord */
    FieldCoord = (double *)malloc(sizeof(double) * Xdots * Ydots * 2);
    if (FieldCoord == NULL) {
        fprintf(stderr, "(Error) >> Cannot allocate FieldCoord[%d,%d,2]\n", Xdots, Ydots);
        exit(-1);
    }
    SetDoubleValue(FieldCoord, Xdots * Ydots * 2,
                   (double)0); // OPP: you can use calloc?

    /* Now read Sreal, Simag, Rreal, Rimag */
    Sreal = Simag = Rreal = Rimag = 0.0;
    while (1) {
        if (readrow(filerow, 80, inpunit) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Sreal from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#')
            continue;
        if (sscanf(filerow, "%lf", &Sreal) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Sreal from string.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Simag) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Simag from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rreal) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Rreal from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rimag) < 1) {
            fprintf(stderr, "(Error) >> Cannot read Rimag from input file.\n");
            exit(-1);
        }
        break;
    }

    /* Now read MaxIters */
    MaxIters = 0;
    while (1) {
        if (readrow(filerow, 80, inpunit) < 1) {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1)
            continue;
        if (sscanf(filerow, "%d", &MaxIters) < 1) {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from string.\n");
            exit(-1);
        }
        break;
    }

    /* Now read TimeSteps */
    TimeSteps = 0;
    while (1) {
        if (readrow(filerow, 80, inpunit) < 1) {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1)
            continue;
        if (sscanf(filerow, "%d", &TimeSteps) < 1) {
            fprintf(stderr, "(Error) >> Cannot read TimeSteps from string.\n");
            exit(-1);
        }
        break;
    }

    fclose(inpunit);
    return;
}

double *FieldDistribution() {
    /*
    !  Compute theoretical value distribution of the perturbing field
    !  Output: TheorSlope(TSlopeLength,3) - theoretical field distribution
    function
    */
    double *CoeffMatrix, *B;
    double x0, y0, x1, y1;
    double t0, t1;

    int M, Mm1, N, Nm1, LA;
    int i, rc;

    fprintf(stdout, "\t>> Computing theoretical perturbing field...\n");

    x0 = Sreal;
    y0 = Simag;
    x1 = x0 + Rreal;
    y1 = y0 + Rimag;

    // How many intervals? It should be safe to use SQRT(Xdots)
    M = sqrt((double)Xdots);
    N = sqrt((double)Ydots);

    Nm1 = N - 1; // Grid points minus boundary
    Mm1 = M - 1; // Grid points minus boundary

    LA = Mm1 * Nm1; // unknown points
    TSlopeLength = LA;

    CoeffMatrix = (double *)malloc(sizeof(double) * LA * LA);
    TheorSlope = (double *)malloc(sizeof(double) * TSlopeLength * 3);
    B = (double *)malloc(sizeof(double) * LA);

    if (CoeffMatrix == NULL || TheorSlope == NULL || B == NULL) {
        fprintf(
            stderr,
            "(Error) >> Cannot allocate memory. \nCoeffMatrix: %p; TheorSlope: "
            "%p, B: %p\n",
            CoeffMatrix, TheorSlope, B);
        exit(-1);
    }

    GridDef(x0, x1, y0, y1, N, TheorSlope);
    EqsDef(x0, x1, y0, y1, N, LA, CoeffMatrix, B, TheorSlope);
    // GridDef(x0, x1, y0, y1, N, TheorSlope);

    rc = LinEquSolve(CoeffMatrix, LA, B);
    if (rc != 0)
        exit(-1);

    for (i = 0; i < LA; i++)
        TheorSlope[index2D(i, 2, TSlopeLength)] = B[i]; // OPP: why not use memcpy?

    free(CoeffMatrix);
    // free(B);

    return B;
}

double *FieldDistribution_mixed() {
    /*
    !  Compute theoretical value distribution of the perturbing field
    !  Output: TheorSlope(TSlopeLength,3) - theoretical field distribution
    function
    */
    double *CoeffMatrix, *B;
    double x0, y0, x1, y1;
    double t0, t1;

    int M, Mm1, N, Nm1, LA;
    int i, rc;

    fprintf(stdout, "\t>> Computing theoretical perturbing field...\n");

    x0 = Sreal;
    y0 = Simag;
    x1 = x0 + Rreal;
    y1 = y0 + Rimag;

    // How many intervals? It should be safe to use SQRT(Xdots)
    M = sqrt((double)Xdots);
    N = sqrt((double)Ydots);

    Nm1 = N - 1; // Grid points minus boundary
    Mm1 = M - 1; // Grid points minus boundary

    LA = Mm1 * Nm1; // unknown points
    TSlopeLength = LA;

    CoeffMatrix = (double *)malloc(sizeof(double) * LA * LA);
    TheorSlope = (double *)malloc(sizeof(double) * TSlopeLength * 3);
    B = (double *)malloc(sizeof(double) * LA);

    if (CoeffMatrix == NULL || TheorSlope == NULL || B == NULL) {
        fprintf(
            stderr,
            "(Error) >> Cannot allocate memory. \nCoeffMatrix: %p; TheorSlope: "
            "%p, B: %p\n",
            CoeffMatrix, TheorSlope, B);
        exit(-1);
    }

    GridDef(x0, x1, y0, y1, N, TheorSlope);
    EqsDef(x0, x1, y0, y1, N, LA, CoeffMatrix, B, TheorSlope);

    double *result_seq = B;
    double *result_acc = NULL;
    result_acc = (double *)malloc(sizeof(double) * LA);
    memcpy(result_acc, B, sizeof(double) * LA);

    t0 = clock();
    double *d_A, *d_b;
    hipMalloc(&d_A, sizeof(double) * LA * LA);
    hipMemcpy(d_A, CoeffMatrix, sizeof(double) * LA * LA, hipMemcpyHostToDevice);
    hipMalloc(&d_b, sizeof(double) * LA);
    hipMemcpy(d_b, B, sizeof(double) * LA, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    rc = LinEquSolve_ACC(d_A, LA, d_b);
    t1 = clock();
    hipMemcpy(result_acc, d_b, sizeof(double) * LA, hipMemcpyDeviceToHost);
    fprintf(stdout, "\t>> LinEquSolve_CUDA took %lf seconds\n", (double)(t1 - t0) / CLOCKS_PER_SEC);
    t0 = clock();
    rc = LinEquSolve(CoeffMatrix, LA, result_seq);
    t1 = clock();
    fprintf(stdout, "\t>> LinEquSolve_seq took %lf seconds\n", (double)(t1 - t0) / CLOCKS_PER_SEC);

    for (i = 0; i < LA; i++) {
        result_seq[i] -= result_acc[i];
    }
    double ninf = -1;
    for (i = 0; i < LA; i++)
        ninf = max(ninf, abs(result_seq[i]));

    printf("---------maximum difference between solutions is %f. Good enough?\n", ninf);
    if (rc != 0)
        exit(-1);

    for (i = 0; i < LA; i++)
        TheorSlope[index2D(i, 2, TSlopeLength)] = B[i]; // OPP: why not use memcpy?

    free(CoeffMatrix);
    // free(B);

    return B;
}

void SensiblePoints(double Ir, double Ii, double Sr, double Si, int MaxIt) {
    /*
    !  Compute "heated" points
    !  Output:
    !          FieldCoord(Xdots,Ydots,2)
    !          FieldWeight(Xdots,Ydots)
   */

    int ix, iy, iz;
    double ca, cb, za, zb;
    double rad, zan, zbn;
    double Xinc, Yinc;

    fprintf(stdout, "\t>> Computing sensitivity to field effects...\n");

    Xinc = Sr / (double)Xdots;
    Yinc = Si / (double)Ydots;

    for (iy = 0; iy < Ydots; iy++) {
        for (ix = 0; ix < Xdots; ix++) {
            ca = Xinc * ix + Ir;
            cb = Yinc * iy + Ii;
            FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)] = ca;
            FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)] = cb;
            rad = ca * ca * ((double)1.0 + (cb / ca) * (cb / ca));
            zan = 0.0;
            zbn = 0.0;
            for (iz = 1; iz <= MaxIt; iz++) {
                if (rad > (double)4.0)
                    break;
                za = zan;
                zb = zbn;
                zan = ca + (za - zb) * (za + zb);
                zbn = 2.0 * (za * zb + cb / 2.0);
                rad = zan * zan * ((double)1.0 + (zbn / zan) * (zbn / zan));
            }
            FieldWeight[index2D(ix, iy, Xdots)] = iz;
        }
    }

    return;
}

void FieldInit() {
    /*
    ! Initialize field values in the grid. Values are computed on the basis
    ! of the measured values read in subroutine InitGrid and the gross grid
    ! values computed in subroutine FieldDistribution. Moreover sensitiveness
    ! to field effects as computed in subroutine SensiblePoints are taken into
    ! account.
    !
    ! Input:
    !        MeasuredValues(:,3)
    !        FieldWeight(Xdots,Ydots)
    ! Output:
    !        FieldValues(Xdots,Ydots,2)
    */

    int rv;
    double xc, yc, ev, sv, sd, DiscrValue;
    double *DiffValues;

    fprintf(stdout, "\t>> Initializing entity of field effects...\n");

    /* Allocate FieldValues */
    FieldValues = (double *)malloc(sizeof(double) * Xdots * Ydots * 2);
    if (FieldValues == NULL) {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate FieldValues[%d,%d,2]\n", Xdots, Ydots);
        exit(-1);
    }
    SetDoubleValue(FieldValues, Xdots * Ydots * 2,
                   (double)0); // OPP: you can use calloc?

    /* Allocate DiffValues */
    DiffValues = (double *)malloc(sizeof(double) * NumInputValues);
    if (DiffValues == NULL) {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate DiffValues[%d]\n", NumInputValues);
        exit(-1);
    }
    SetDoubleValue(DiffValues, NumInputValues,
                   (double)0.0); // OPP: you can use calloc?

    /* Compute discrepancy between Measured and Theoretical value */
    DiscrValue = 0.0;
    for (rv = 0; rv < NumInputValues; rv++) {
        xc = MeasuredValues[index2D(rv, 0, NumInputValues)];
        yc = MeasuredValues[index2D(rv, 1, NumInputValues)];

        // TheorSlope is computed on the basis of a coarser grid, so look for the
        // best values near xc, yc coordinates
        sv = NearestValue(xc, yc, TSlopeLength, TheorSlope);
        ev = MeasuredValues[index2D(rv, 2, NumInputValues)];

        DiffValues[rv] = ev - sv;
        DiscrValue += ev - sv;
    }
    DiscrValue = DiscrValue / (double)NumInputValues;

    // Compute standard deviation
    sd = 0.0;
    for (rv = 0; rv < NumInputValues; rv++)
        sd = sd + (DiffValues[rv] - DiscrValue) * (DiffValues[rv] - DiscrValue);
    sd = sqrt(sd / (double)NumInputValues);

    // Print statistics
    fprintf(
        stdout,
        "\t...Number of Points, Mean value, Standard deviation = %d, %12.3e, "
        "%12.3e\n",
        NumInputValues, DiscrValue, sd);

    // Compute FieldValues stage 1
    FieldPoints(DiscrValue);

    free(DiffValues);

    return;
}

void Cooling(int steps) {
    /*
    !  Compute evolution of the effects of the field
    !  Input/Output:
    !                FieldValues(Xdots,Ydots,2)
    */

    int iz, it;
    char fname[80];
    double vmin, vmax;

    fprintf(stdout, "\t>> Computing cooling of field effects ...\n");
    fprintf(stdout, "\t... %d steps ...\n", steps);
    sprintf(fname, "FieldValues0000");

    vmin = vmax = 0.0;
    // RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], &vmin, &vmax, fname);
    Statistics(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], 0);

    iz = 1;
    for (it = 1; it <= steps; it++) {
        // Update the value of grid points
        Update(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)],
               &FieldValues[index3D(0, 0, 2 - iz, Xdots, Ydots)]);
        iz = 3 - iz;

        // Print and show results
        sprintf(fname, "FieldValues%4.4d", it);
        // if (it % 4 == 0) RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &vmin, &vmax, fname);
        Statistics(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], it);
    }

    return;
}

void GridDef(double x0, double x1, double y0, double y1, int N, double *Pts) {
    double x, y, dx, dy;
    int i, j, np, Mm1, Nm1;

    Mm1 = sqrt((double)Xdots) - 1;
    Nm1 = sqrt((double)Ydots) - 1;
    dx = (x1 - x0) / (double)N;
    dy = (y1 - y0) / (double)N;

    np = -1;
    for (i = 0; i < Mm1; i++) {
        for (j = 0; j < Nm1; j++) {
            np++;
            if (np > Mm1 * Nm1) {
                fprintf(stderr, "(Error@GridDef) >> NP = %d > N*N = %d\n", np, Nm1 * Nm1);
                exit(-1);
            }
            x = x0 + dx * (double)(i + 1);
            y = y0 + dy * (double)(j + 1);
            Pts[index2D(np, 0, TSlopeLength)] = x;
            Pts[index2D(np, 1, TSlopeLength)] = y;
        }
    }
    return;
}

void EqsDef(double x0, double x1, double y0, double y1, int N, int LA, double *A, double *Rhs, double *Pts) {
    // Pts(LA,3) - inner grid point Coordinates
    // Rhs(LA)   - Linear equation Right Hand Side
    // A(LA,LA)  - Linear equation matrix

    double x, y, Eps, dx, dy;
    int np, Nm1, pos;

    //  Define A matrix and RHS

    Nm1 = N - 1;
    dx = (x1 - x0) / (double)N;
    dy = (y1 - y0) / (double)N;

    SetDoubleValue(A, LA * LA, (double)0); // OPP: you can use calloc?
    SetDoubleValue(Rhs, LA, (double)0);    // OPP: you can use calloc?

    for (np = 0; np < LA; np++) {
        x = Pts[index2D(np, 0, TSlopeLength)];
        y = Pts[index2D(np, 1, TSlopeLength)];

        A[index2D(np, np, LA)] = -4.0;

        Rhs[np] = (x + y) * dx * dy;

        // define Eps function of grid dimensions
        Eps = (dx + dy) / 20.0;

        // where is P(x-dx,y) ?
        if (fabs((x - dx) - x0) < Eps)
            Rhs[np] = Rhs[np] - Solution(x0, y);
        else {
            // Find pos = position of P(x-dx,y)
            pos = np - Nm1;
            if (fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x - dx)) > Eps) {
                fprintf(stderr,
                        "(Error@EqsDef) >> x-dx: pos, np, d = %d %d %lf\n", pos,
                        np,
                        fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x - dx)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x+dx,y) ?
        if (fabs((x + dx) - x1) < Eps)
            Rhs[np] = Rhs[np] - Solution(x1, y);
        else {
            // Find pos = position of P(x+dx,y)
            pos = np + Nm1;
            if (fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x + dx)) > Eps) {
                fprintf(stderr, "(Error@EqsDef) >> x+dx: %lf\n",
                        fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x + dx)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y-dy) ?
        if (fabs((y - dy) - y0) < Eps)
            Rhs[np] = Rhs[np] - Solution(x, y0);
        else {
            // Find pos = position of P(x,y-dy)
            pos = np - 1;
            if (fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y - dy)) > Eps) {
                fprintf(stderr, "(Error@EqsDef) >> y-dy: %lf\n",
                        fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y - dy)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y+dy) ?
        if (fabs((y + dy) - y1) < Eps)
            Rhs[np] = Rhs[np] - Solution(x, y1);
        else {
            // Find pos = position of P(x,y-dy)
            pos = np + 1;
            if (fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y + dy)) > Eps) {
                fprintf(stderr, "(Error@EqsDef) >> y+dy: %lf\n",
                        fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y + dy)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }
    }
    return;
}

double Solution(double x, double y) {
    return ((x * x * x) + (y * y * y)) / (double)6.0;
}

int LinEquSolve(double *a, int n, double *b) {
    /* Gauss-Jordan elimination algorithm */
    int i, j, k, l, icol, irow;
    int *indcol, *indrow, *ipiv;
    double bigger, temp;

    /* Allocate indcol */
    indcol = (int *)malloc(sizeof(int) * n);
    if (indcol == NULL) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indcol[%d]\n", n);
        return (-1);
    }

    /* Allocate indrow */
    indrow = (int *)malloc(sizeof((int)1) * n);
    if (indrow == NULL) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indrow[%d]\n", n);
        return (-1);
    }

    /* Allocate ipiv */
    ipiv = (int *)malloc(sizeof((int)1) * n);
    if (ipiv == NULL) {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate ipiv[%d]\n", n);
        return (-1);
    }
    SetIntValue(ipiv, n, 0); // OPP: you can use calloc?

    /* Actual algorithm */

    for (i = 0; i < n; i++) {
        bigger = 0.0;

        for (j = 0; j < n; j++) {
            if (ipiv[j] != 1) {
                for (k = 0; k < n; k++) {
                    if (ipiv[k] == 0 && bigger <= fabs(a[index2D(j, k, n)])) {
                        bigger = fabs(a[index2D(j, k, n)]);
                        irow = j;
                        icol = k;
                    }
                }
            }
        }

        ipiv[icol] = ipiv[icol] + 1;

        if (irow != icol) {
            for (l = 0; l < n; l++) {
                temp = a[index2D(irow, l, n)];
                a[index2D(irow, l, n)] = a[index2D(icol, l, n)];
                a[index2D(icol, l, n)] = temp;
            }
            temp = b[irow];
            b[irow] = b[icol];
            b[icol] = temp;
        }

        indrow[i] = irow;
        indcol[i] = icol;

        if (a[index2D(icol, icol, n)] == 0.0) {
            fprintf(stderr, "(Error@LinEquSolve) >> a(%d,%d): singular matrix!", icol, icol);
            return -2;
        }

        temp = (double)1.0 / a[index2D(icol, icol, n)];
        a[index2D(icol, icol, n)] = 1.0;

        for (l = 0; l < n; l++)
            a[index2D(icol, l, n)] = a[index2D(icol, l, n)] * temp;

        b[icol] = b[icol] * temp;

        for (l = 0; l < n; l++) {
            if (l != icol) {
                temp = a[index2D(l, icol, n)];
                a[index2D(l, icol, n)] = 0.0;
                for (k = 0; k < n; k++) {
                    a[index2D(l, k, n)] = a[index2D(l, k, n)] - a[index2D(icol, k, n)] * temp;
                }
                b[l] = b[l] - b[icol] * temp;
            }
        }
    }

    for (l = n - 1; l >= 0; l--) {
        if (indrow[l] != indcol[l]) {
            for (k = 0; k < n; k++) {
                temp = a[index2D(k, indrow[l], n)];
                a[index2D(k, indrow[l], n)] = a[index2D(k, indcol[l], n)];
                a[index2D(k, indcol[l], n)] = temp;
            }
        }
    }

    free(indcol);
    free(indrow);
    free(ipiv);

    return 0;
}

double NearestValue(double xc, double yc, int ld, double *Values) {

    // look for the best values near xc, yc coordinates
    double v;

    double d, md; // minimum distance
    int np;       // number of nearest points
    int i;

    md = ((xc - Values[index2D(0, 0, ld)]) * (xc - Values[index2D(0, 0, ld)])) + ((yc - Values[index2D(0, 1, ld)]) * (yc - Values[index2D(0, 1, ld)]));

    // Compute lowest distance
    for (i = 0; i < ld; i++) {
        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));
        if (md > d)
            md = d;
    }

    np = 0;
    v = 0.0;

    // Compute nearest value
    for (i = 0; i < ld; i++) {
        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));
        if (md == d) {
            // add contributed value
            np = np + 1;
            v = v + Values[index2D(i, 2, ld)];
        }
    }

    // mean value
    v = v / (double)np;

    return v;
}

void FieldPoints(double Diff) {
    int ix, iy;
    double xc, yc, sv;
    int rmin, rmax;

    rmax = MaxIntVal(Xdots * Ydots, FieldWeight);
    rmin = MinIntVal(Xdots * Ydots, FieldWeight);

    printf("-----> rmin = %d, rmax = %d on CPU\n", rmin, rmax);

    for (iy = 0; iy < Ydots; iy++) {
        for (ix = 0; ix < Xdots; ix++) {
            xc = FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)];
            yc = FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)];

            // Compute effects of field in every point
            sv = NearestValue(xc, yc, TSlopeLength, TheorSlope);
            FieldValues[index3D(ix, iy, 0, Xdots, Ydots)] = 293.16 + 80 * (Diff + sv) * (FieldWeight[index2D(ix, iy, Xdots)] - rmin) / (rmax - rmin);
        }
    }

    // Copy initial status
    // OPP: use memcpy?
    for (iy = 0; iy < Ydots; iy++) {
        for (ix = 0; ix < Xdots; ix++) {
            FieldValues[index3D(ix, iy, 1, Xdots, Ydots)] = FieldValues[index3D(ix, iy, 0, Xdots, Ydots)];
        }
    }

    return;
}

void Update(int xdots, int ydots, double *u1, double *u2) {
    /* Compute next step using matrices g1, g2 of dimension (nr,nc) */

    int i, j;
    double CX, CY;
    double hx, dgx, hy, dgy, dd;

    dd = 0.0000001;
    hx = 1.0 / (double)xdots;
    hy = 1.0 / (double)ydots;
    dgx = -2.0 + hx * hx / (2 * dd);
    dgy = -2.0 + hy * hy / (2 * dd);
    CX = dd / (hx * hx);
    CY = dd / (hy * hy);

    for (j = 0; j < ydots - 1; j++) {
        for (i = 0; i < xdots - 1; i++) {
            if (i <= 0 || i >= xdots - 1) {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            if (j <= 0 || j >= ydots - 1) {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            u2[index2D(i, j, xdots)] = CX * (u1[index2D((i - 1), j, xdots)] + u1[index2D((i + 1), j, xdots)] + dgx * u1[index2D((i + 1), j, xdots)]) + CY * (u1[index2D(i, (j - 1), xdots)] + u1[index2D(i, (j + 1), xdots)] + dgy * u1[index2D(i, j, xdots)]);
        }
    }

    for (j = 0; j < ydots - 1; j++) {
        u2[index2D(0, j, xdots)] = u2[index2D(1, j, xdots)];
        u2[index2D(Xdots - 1, j, xdots)] = u2[index2D(Xdots - 2, j, xdots)];
    }

    for (i = 0; i < xdots - 1; i++) {
        u2[index2D(i, 0, xdots)] = u2[index2D(i, 1, xdots)];
        u2[index2D(i, Ydots - 1, xdots)] = u2[index2D(i, Ydots - 2, xdots)];
    }

    return;
}
