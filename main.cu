#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include "util.cu"

#define index2D(i,j,LD1) i + ((j)*LD1)    // element position in 2-D arrays
#define index3D(i,j,k,LD1,LD2) i + ((j)*LD1) + ((k)*LD1*LD2)   // element position in 3-D arrays

#define Xdots 1400   // Plate grid resolution in 2 dimensions
#define Ydots 1400   // May be changed to 1000x1000

// Parameters to compute point sensitiveness - values read from input file
double Sreal, Simag, Rreal, Rimag;
int MaxIters;
int TimeSteps;   // Evolution time steps

double* MeasuredValues;    // 2-D array - (NumInputValues,3) - Values read in input file
int NumInputValues;        // Number of values read in input file
double* TheorSlope;        // 2-D array - Theoretical value distribution
int TSlopeLength;          // TheorSlope grid dimensions
int* FieldWeight;          // 2-D array - (Xdots,Ydots) - Degree of sensitiveness to perturbing field 
double* FieldCoord;        // 3-D array - X, Y coordinates in field
double* FieldValues;       // 3-D array - X, Y coordinates in field

/* CODE */

//  functions  prototypes

    void InitGrid(char* InputFile);
    int LinEquSolve(double* a, int n, double* b);
    void EqsDef(double x0, double x1, double y0, double y1, int N, int LA, double* A, double* Rhs, double* Pts);
    double Solution(double x, double y);
    void FieldDistribution();
    void GridDef(double x0, double x1, double y0, double y1, int N, double* Pts);
    void SensiblePoints(double Ir, double Ii, double Sr, double Si, int MaxIt);
    void FieldInit();
    double NearestValue(double xc, double yc, int ld, double* Values);
    void FieldPoints(double Diff);
    void Cooling(int steps);
    void RealData2ppm(int s1, int s2, double* rdata, double* vmin, double* vmax, char* name);
    void Statistics(int s1, int s2, double* rdata, int s);
    void Update(int xdots, int ydots, double* u1, double* u2);

int main(int argc, char* argv[])
{
    clock_t t0, t1, p0, p1;

    t0 = clock();
    printf(">> Starting\n");

    // Read input file
    p0 = clock();
    InitGrid("Cooling.inp");
    p1 = clock();
    fprintf(stdout, ">> InitGrid ended in %lf seconds\n", (double)(p1 - p0)/CLOCKS_PER_SEC);

    // TheorSlope(TSlopeLength,3)
    p0 = clock();
    FieldDistribution();
    p1 = clock();
    fprintf(stdout, ">> FieldDistribution ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    // FieldCoord(Xdots,Ydots,2), FieldWeight(Xdots,Ydots)
    p0 = clock();
    SensiblePoints(Sreal, Simag, Rreal, Rimag, MaxIters);
    p1 = clock();
    fprintf(stdout, ">> SensiblePoints ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    // MeasuredValues(:,3), FieldWeight(Xdots,Ydots) -> FieldValues(Xdots,Ydots,2)
    p0 = clock();
    FieldInit();
    p1 = clock();
    fprintf(stdout, ">> FieldInit ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    // FieldValues(Xdots,Ydots,2)
    p0 = clock();
    Cooling(TimeSteps);
    p1 = clock();
    fprintf(stdout, ">> Cooling ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    t1 = clock();
    fprintf(stdout, ">> Computations ended in %lf seconds\n", (double)(t1 - t0) / CLOCKS_PER_SEC);

    // End Program

    free(MeasuredValues);
    free(FieldWeight);
    free(FieldCoord);
    free(TheorSlope);
    free(FieldValues);

    return 0;

    /*time_t t0, t1, p0, p1;

    time(&t0);
    //fprintf(stdout, ">> Starting %s at: %s", argv[0], asctime(localtime(&t0)));
    printf(">> Starting\n");

    // Read input file
    time(&p0);
    InitGrid("Cooling.inp");
    time(&p1);
    fprintf(stdout, ">> InitGrid ended in %lf seconds\n", difftime(p1, p0));

    // TheorSlope(TSlopeLength,3)
    time(&p0);
    FieldDistribution();
    time(&p1);
    fprintf(stdout, ">> FieldDistribution ended in %lf seconds\n", difftime(p1, p0));

    // FieldCoord(Xdots,Ydots,2), FieldWeight(Xdots,Ydots)
    time(&p0);
    SensiblePoints(Sreal, Simag, Rreal, Rimag, MaxIters);
    time(&p1);
    fprintf(stdout, ">> SensiblePoints ended in %lf seconds\n", difftime(p1, p0));

    // MeasuredValues(:,3), FieldWeight(Xdots,Ydots) -> FieldValues(Xdots,Ydots,2)
    time(&p0);
    FieldInit();
    time(&p1);
    fprintf(stdout, ">> FieldInit ended in %lf seconds\n", difftime(p1, p0));

    // FieldValues(Xdots,Ydots,2)
    time(&p0);
    Cooling(TimeSteps);
    time(&p1);
    fprintf(stdout, ">> Cooling ended in %lf seconds\n", difftime(p1, p0));

    time(&t1);
    fprintf(stdout, ">> Ending at: %s", asctime(localtime(&t1)));
    fprintf(stdout, ">> Computations ended in %lf seconds\n", difftime(t1, t0));

    // End Program

    free(MeasuredValues);
    free(FieldWeight);
    free(FieldCoord);
    free(TheorSlope);
    free(FieldValues);

	return 0;*/
}

/* FUNCTIONS */

void InitGrid(char* InputFile)
{
    /* Output:
    !  MeasuredValues(:,3) - values read from input file
    !  Initialization of FieldWeight(Xdots,Ydots) and FieldCoord(Xdots,Ydots,2)
    */

    int valrows, st;
    char filerow[80];
    FILE* inpunit;

    fprintf(stdout, ">> Initializing grid ...\n");

    inpunit = fopen(InputFile, "r");
    if (!inpunit) 
    {
        fprintf(stderr, "(Error) >>> Cannot access file %s\n", InputFile);
        exit(-1);
    }

    // Read measured values
    NumInputValues = 0;
    valrows = 0;
    while (1)
    {
        st = readrow(filerow, 80, inpunit);
        if (filerow[0] == '#') continue;
        if (NumInputValues <= 0) 
        {
            if (sscanf(filerow, "  %d", &NumInputValues) < 1) 
            {
                if (NumInputValues <= 0) 
                {
                    fprintf(stderr, "(Error) >> there seems to be %d input values...\n", NumInputValues);
                    exit(-1);
                }
            }
            else 
            {
                MeasuredValues = (double*)malloc(sizeof(double) * NumInputValues * 3);
                if (MeasuredValues == NULL) 
                {
                    fprintf(stderr, "(Error) >> Cannot allocate MeasuredValues[%d,3] :(\n", NumInputValues);
                    exit(-1);
                }
            }
        }
        else 
        {
            if (sscanf(filerow, "%lf %lf %lf",
                &MeasuredValues[index2D(valrows, 0, NumInputValues)],  // X coord
                &MeasuredValues[index2D(valrows, 1, NumInputValues)],  // Y coord
                &MeasuredValues[index2D(valrows, 2, NumInputValues)])  // Measured value
                < 3) 
            {
                fprintf(stderr, "(Error) >>> something went wrong while reading MeasuredValues(%d,*)", valrows);
                exit(-1);
            }
            valrows++;
            if (valrows >= NumInputValues) break;
        }
    }

    /* Create and initialize FieldWeight */
    FieldWeight = (int*)malloc(sizeof(int) * Xdots * Ydots);
    if (FieldWeight == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldWeight[%d,%d]\n", Xdots, Ydots);
        exit(-1);
    }
    SetIntValue(FieldWeight, Xdots * Ydots, 0); // OPP: you can use calloc?

    /* Create and initialize FieldCoord */
    FieldCoord = (double*)malloc(sizeof(double) * Xdots * Ydots * 2);
    if (FieldCoord == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldCoord[%d,%d,2]\n", Xdots, Ydots);
        exit(-1);
    }
    SetDoubleValue(FieldCoord, Xdots * Ydots * 2, (double)0); // OPP: you can use calloc?

    /* Now read Sreal, Simag, Rreal, Rimag */
    Sreal = Simag = Rreal = Rimag = 0.0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Sreal from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#') continue;
        if (sscanf(filerow, "%lf", &Sreal) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Sreal from string.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Simag) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Simag from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rreal) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Rreal from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rimag) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Rimag from input file.\n");
            exit(-1);
        }
        break;
    }

    /* Now read MaxIters */
    MaxIters = 0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1) continue;
        if (sscanf(filerow, "%d", &MaxIters) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from string.\n");
            exit(-1);
        }
        break;
    }

    /* Now read TimeSteps */
    TimeSteps = 0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1) continue;
        if (sscanf(filerow, "%d", &TimeSteps) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read TimeSteps from string.\n");
            exit(-1);
        }
        break;
    }

    fclose(inpunit);
    return;
}

void FieldDistribution()
{
    /*
    !  Compute theoretical value distribution of the perturbing field
    !  Output: TheorSlope(TSlopeLength,3) - theoretical field distribution function
    */
    double *CoeffMatrix, *B;
    double x0, y0, x1, y1;

    int M, Mm1, N, Nm1, LA;
    int i, rc;

    fprintf(stdout, "\t>> Computing theoretical perturbing field...\n");

    x0 = Sreal; 
    y0 = Simag; 
    x1 = x0 + Rreal; 
    y1 = y0 + Rimag;

    // How many intervals? It should be safe to use SQRT(Xdots)
    M = sqrt((double)Xdots);
    N = sqrt((double)Ydots);

    Nm1 = N - 1;  // Grid points minus boundary
    Mm1 = M - 1;  // Grid points minus boundary

    LA = Mm1 * Nm1; // unknown points
    TSlopeLength = LA;

    /* Allocate CoeffMatrix */
    CoeffMatrix = (double*)malloc(sizeof(double) * LA * LA);
    if (CoeffMatrix == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate CoeffMatrix[%d,%d]\n", LA, LA);
        exit(-1);
    }

    /* Allocate TheorSlope */
    TheorSlope = (double*)malloc(sizeof(double) * TSlopeLength * 3);
    if (TheorSlope == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate TheorSlope[%d,3]\n", TSlopeLength);
        exit(-1);
    }

    /* Allocate B */
    B = (double*)malloc(sizeof(double) * LA);
    if (B == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate B[%d]\n", LA);
        exit(-1);
    }

    GridDef(x0, x1, y0, y1, N, TheorSlope);

    EqsDef(x0, x1, y0, y1, N, LA, CoeffMatrix, B, TheorSlope);

    rc = LinEquSolve(CoeffMatrix, LA, B);
    if (rc != 0) exit(-1);

    for (i = 0; i < LA; i++) TheorSlope[index2D(i, 2, TSlopeLength)] = B[i]; // OPP: why not use memcpy?

    free(CoeffMatrix);
    free(B);

    return;
}

void SensiblePoints(double Ir, double Ii, double Sr, double Si, int MaxIt)
{
    /*
    !  Compute "heated" points
    !  Output:
    !          FieldCoord(Xdots,Ydots,2)
    !          FieldWeight(Xdots,Ydots)
   */

    int ix, iy, iz;
    double ca, cb, za, zb;
    double rad, zan, zbn;
    double Xinc, Yinc;

    fprintf(stdout, "\t>> Computing sensitivity to field effects...\n");

    Xinc = Sr / (double)Xdots;
    Yinc = Si / (double)Ydots;

    for (iy = 0; iy < Ydots; iy++) 
    {
        for (ix = 0; ix < Xdots; ix++) 
        {
            ca = Xinc * ix + Ir;
            cb = Yinc * iy + Ii;
            FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)] = ca;
            FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)] = cb;
            rad = ca * ca * ((double)1.0 + (cb / ca) * (cb / ca));
            zan = 0.0;
            zbn = 0.0;
            for (iz = 1; iz <= MaxIt; iz++) 
            {
                if (rad > (double)4.0) break;
                za = zan;
                zb = zbn;
                zan = ca + (za - zb) * (za + zb);
                zbn = 2.0 * (za * zb + cb / 2.0);
                rad = zan * zan * ((double)1.0 + (zbn / zan) * (zbn / zan));
            }
            FieldWeight[index2D(ix, iy, Xdots)] = iz;
        }
    }

    return;
}

void FieldInit()
{
    /*
    ! Initialize field values in the grid. Values are computed on the basis
    ! of the measured values read in subroutine InitGrid and the gross grid
    ! values computed in subroutine FieldDistribution. Moreover sensitiveness
    ! to field effects as computed in subroutine SensiblePoints are taken into
    ! account.
    !
    ! Input:
    !        MeasuredValues(:,3)
    !        FieldWeight(Xdots,Ydots)
    ! Output:
    !        FieldValues(Xdots,Ydots,2)
    */

    int rv;
    double xc, yc, ev, sv, sd, DiscrValue;
    double *DiffValues;

    fprintf(stdout, "\t>> Initializing entity of field effects...\n");

    /* Allocate FieldValues */
    FieldValues = (double*)malloc(sizeof(double) * Xdots * Ydots * 2);
    if (FieldValues == NULL) 
    {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate FieldValues[%d,%d,2]\n", Xdots, Ydots);
        exit(-1);
    }
    SetDoubleValue(FieldValues, Xdots * Ydots * 2, (double)0); // OPP: you can use calloc?

    /* Allocate DiffValues */
    DiffValues = (double*)malloc(sizeof(double) * NumInputValues);
    if (DiffValues == NULL) 
    {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate DiffValues[%d]\n", NumInputValues);
        exit(-1);
    }
    SetDoubleValue(DiffValues, NumInputValues, (double)0.0); // OPP: you can use calloc?

    /* Compute discrepancy between Measured and Theoretical value */
    DiscrValue = 0.0;
    for (rv = 0; rv < NumInputValues; rv++) 
    {
        xc = MeasuredValues[index2D(rv, 0, NumInputValues)];
        yc = MeasuredValues[index2D(rv, 1, NumInputValues)];

        // TheorSlope is computed on the basis of a coarser grid, so look for the best values near xc, yc coordinates
        sv = NearestValue(xc, yc, TSlopeLength, TheorSlope);
        ev = MeasuredValues[index2D(rv, 2, NumInputValues)];

        DiffValues[rv] = ev - sv;
        DiscrValue += ev - sv;
    }
    DiscrValue = DiscrValue / (double)NumInputValues;

    // Compute standard deviation
    sd = 0.0;
    for (rv = 0; rv < NumInputValues; rv++) sd = sd + (DiffValues[rv] - DiscrValue) * (DiffValues[rv] - DiscrValue);
    sd = sqrt(sd / (double)NumInputValues);

    // Print statistics
    fprintf(stdout, "\t...Number of Points, Mean value, Standard deviation = %d, %12.3e, %12.3e\n", NumInputValues, DiscrValue, sd);

    // Compute FieldValues stage 1
    FieldPoints(DiscrValue);

    free(DiffValues);

    return;
}

void Cooling(int steps)
{
    /*
    !  Compute evolution of the effects of the field
    !  Input/Output:
    !                FieldValues(Xdots,Ydots,2)
    */

    int iz, it;
    char fname[80];
    double vmin, vmax;

    fprintf(stdout, "\t>> Computing cooling of field effects ...\n");
    fprintf(stdout, "\t... %d steps ...\n", steps);
    sprintf(fname, "FieldValues0000");

    vmin = vmax = 0.0;
    //RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], &vmin, &vmax, fname);
    Statistics(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], 0);

    iz = 1;
    for (it = 1; it <= steps; it++) 
    {
        // Update the value of grid points
        Update(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &FieldValues[index3D(0, 0, 2 - iz, Xdots, Ydots)]);
        iz = 3 - iz;

        // Print and show results 
        sprintf(fname, "FieldValues%4.4d", it);
        //if (it % 4 == 0) RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &vmin, &vmax, fname);
        Statistics(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], it);
    }

    return;
}

/* SUB-FUNCTIONS */

void GridDef(double x0, double x1, double y0, double y1, int N, double* Pts)
{
    double x, y, dx, dy;
    int i, j, np, Mm1, Nm1;

    Mm1 = sqrt((double)Xdots) - 1;
    Nm1 = sqrt((double)Ydots) - 1;
    dx = (x1 - x0) / (double)N; 
    dy = (y1 - y0) / (double)N;

    np = -1;
    for (i = 0; i < Mm1; i++) 
    {
        for (j = 0; j < Nm1; j++) 
        {
            np++;
            if (np > Mm1 * Nm1) 
            {
                fprintf(stderr, "(Error@GridDef) >> NP = %d > N*N = %d\n", np, Nm1 * Nm1);
                exit(-1);
            }
            x = x0 + dx * (double)(i + 1);
            y = y0 + dy * (double)(j + 1);
            Pts[index2D(np, 0, TSlopeLength)] = x;
            Pts[index2D(np, 1, TSlopeLength)] = y;
        }
    }
    return;
}

void EqsDef(double x0, double x1, double y0, double y1, int N, int LA, double* A, double* Rhs, double* Pts)
{
    // Pts(LA,3) - inner grid point Coordinates
    // Rhs(LA)   - Linear equation Right Hand Side
    // A(LA,LA)  - Linear equation matrix

    double x, y, Eps, dx, dy;
    int np, Nm1, pos;

    //  Define A matrix and RHS

    Nm1 = N - 1;
    dx = (x1 - x0) / (double)N; dy = (y1 - y0) / (double)N;

    SetDoubleValue(A, LA * LA, (double)0); // OPP: you can use calloc?
    SetDoubleValue(Rhs, LA, (double)0); // OPP: you can use calloc?

    for (np = 0; np < LA; np++) 
    {
        x = Pts[index2D(np, 0, TSlopeLength)];
        y = Pts[index2D(np, 1, TSlopeLength)];

        A[index2D(np, np, LA)] = -4.0;

        Rhs[np] = (x + y) * dx * dy;

        // define Eps function of grid dimensions 
        Eps = (dx + dy) / 20.0;

        // where is P(x-dx,y) ? 
        if (fabs((x - dx) - x0) < Eps) Rhs[np] = Rhs[np] - Solution(x0, y);
        else 
        {
            // Find pos = position of P(x-dx,y)
            pos = np - Nm1;
            if (fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x - dx)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> x-dx: pos, np, d = %d %d %lf\n", pos, np, fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x - dx)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x+dx,y) ? 
        if (fabs((x + dx) - x1) < Eps) Rhs[np] = Rhs[np] - Solution(x1, y);
        else 
        {
            // Find pos = position of P(x+dx,y)
            pos = np + Nm1;
            if (fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x + dx)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> x+dx: %lf\n", fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x + dx)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y-dy) ? 
        if (fabs((y - dy) - y0) < Eps) Rhs[np] = Rhs[np] - Solution(x, y0);
        else 
        {
            // Find pos = position of P(x,y-dy)
            pos = np - 1;
            if (fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y - dy)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> y-dy: %lf\n", fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y - dy)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y+dy) ? 
        if (fabs((y + dy) - y1) < Eps) Rhs[np] = Rhs[np] - Solution(x, y1);
        else 
        {
            // Find pos = position of P(x,y-dy)
            pos = np + 1;
            if (fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y + dy)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> y+dy: %lf\n", fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y + dy)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }
    }
    return;
}

double Solution(double x, double y)
{
    return ((x * x * x) + (y * y * y)) / (double)6.0;
}

int LinEquSolve(double* a, int n, double* b)
{
    /* Gauss-Jordan elimination algorithm */
    int i, j, k, l, icol, irow;
    int *indcol, *indrow, *ipiv;
    double bigger, temp;

    /* Allocate indcol */
    indcol = (int*)malloc(sizeof(int) * n);
    if (indcol == NULL) 
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indcol[%d]\n", n);
        return(-1);
    }

    /* Allocate indrow */
    indrow = (int*)malloc(sizeof((int)1) * n);
    if (indrow == NULL) 
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indrow[%d]\n", n);
        return(-1);
    }

    /* Allocate ipiv */
    ipiv = (int*)malloc(sizeof((int)1) * n);
    if (ipiv == NULL) 
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate ipiv[%d]\n", n);
        return(-1);
    }
    SetIntValue(ipiv, n, 0); // OPP: you can use calloc?

    /* Actual algorithm */

    for (i = 0; i < n; i++) 
    {
        bigger = 0.0;

        for (j = 0; j < n; j++) 
        {
            if (ipiv[j] != 1) 
            {
                for (k = 0; k < n; k++) 
                {
                    if (ipiv[k] == 0 && bigger <= fabs(a[index2D(j, k, n)])) 
                    {
                        bigger = fabs(a[index2D(j, k, n)]);
                        irow = j;
                        icol = k;
                    }
                }
            }
        }

        ipiv[icol] = ipiv[icol] + 1;

        if (irow != icol) 
        {
            for (l = 0; l < n; l++) 
            {
                temp = a[index2D(irow, l, n)];
                a[index2D(irow, l, n)] = a[index2D(icol, l, n)];
                a[index2D(icol, l, n)] = temp;
            }
            temp = b[irow];
            b[irow] = b[icol];
            b[icol] = temp;
        }

        indrow[i] = irow;
        indcol[i] = icol;

        if (a[index2D(icol, icol, n)] == 0.0) 
        {
            fprintf(stderr, "(Error@LinEquSolve) >> a(%d,%d): singular matrix!", icol, icol);
            return -2;
        }

        temp = (double)1.0 / a[index2D(icol, icol, n)];
        a[index2D(icol, icol, n)] = 1.0;

        for (l = 0; l < n; l++) a[index2D(icol, l, n)] = a[index2D(icol, l, n)] * temp;

        b[icol] = b[icol] * temp;

        for (l = 0; l < n; l++) 
        {
            if (l != icol) 
            {
                temp = a[index2D(l, icol, n)];
                a[index2D(l, icol, n)] = 0.0;
                for (k = 0; k < n; k++) 
                {
                    a[index2D(l, k, n)] = a[index2D(l, k, n)] - a[index2D(icol, k, n)] * temp;
                }
                b[l] = b[l] - b[icol] * temp;
            }
        }
    }

    for (l = n - 1; l >= 0; l--) 
    {
        if (indrow[l] != indcol[l]) 
        {
            for (k = 0; k < n; k++) 
            {
                temp = a[index2D(k, indrow[l], n)];
                a[index2D(k, indrow[l], n)] = a[index2D(k, indcol[l], n)];
                a[index2D(k, indcol[l], n)] = temp;
            }
        }
    }

    free(indcol);
    free(indrow);
    free(ipiv);

    return 0;
}

double NearestValue(double xc, double yc, int ld, double* Values)
{
    // look for the best values near xc, yc coordinates
    double v;

    double d, md; // minimum distance
    int np; // number of nearest points
    int i;

    md = ((xc - Values[index2D(0, 0, ld)]) * (xc - Values[index2D(0, 0, ld)])) +
         ((yc - Values[index2D(0, 1, ld)]) * (yc - Values[index2D(0, 1, ld)]));

    // Compute lowest distance
    for (i = 0; i < ld; i++) 
    {
        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));
        if (md > d) md = d;
    }

    np = 0;
    v = 0.0;

    // Compute nearest value
    for (i = 0; i < ld; i++) 
    {
        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));
        if (md == d) 
        {
            // add contributed value
            np = np + 1;
            v = v + Values[index2D(i, 2, ld)];
        }
    }

    // mean value
    v = v / (double)np;

    return v;
}

void FieldPoints(double Diff)
{
    int ix, iy;
    double xc, yc, sv;
    double rmin, rmax;

    rmax = MaxIntVal(Xdots * Ydots, FieldWeight);
    rmin = MinIntVal(Xdots * Ydots, FieldWeight);

    for (iy = 0; iy < Ydots; iy++) 
    {
        for (ix = 0; ix < Xdots; ix++) 
        {
            xc = FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)];
            yc = FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)];

            // Compute effects of field in every point
            sv = NearestValue(xc, yc, TSlopeLength, TheorSlope);
            FieldValues[index3D(ix, iy, 0, Xdots, Ydots)] = 293.16 + 80 * (Diff + sv) * (FieldWeight[index2D(ix, iy, Xdots)] - rmin) / (rmax - rmin);
        }
    }

    // Copy initial status 
    // OPP: use memcpy?
    for (iy = 0; iy < Ydots; iy++) 
    {
        for (ix = 0; ix < Xdots; ix++) 
        {
            FieldValues[index3D(ix, iy, 1, Xdots, Ydots)] = FieldValues[index3D(ix, iy, 0, Xdots, Ydots)];
        }
    }

    return;
}

void RealData2ppm(int s1, int s2, double* rdata, double* vmin, double* vmax, char* name)
{
    /* Simple subroutine to dump integer data in a PPM format */

    int cm[3][256];  /* R,G,B, Colour Map */
    FILE* ouni, * ColMap;
    int i, j, rc, vp, vs;
    double  rmin, rmax;
    char  fname[80], jname[80], command[80];

    /* Load color map: 256 colours */
    ColMap = fopen("ColorMap.txt", "r");
    if (ColMap == NULL) 
    {
        fprintf(stderr, "(Error@RealData2ppm) >> Cannot open ColorMap.txt\n");
        exit(-1);
    }
    for (i = 0; i < 256; i++) 
    {
        if (fscanf(ColMap, " %3d %3d %3d", &cm[0][i], &cm[1][i], &cm[2][i]) < 3) 
        {
            fprintf(stderr, "(Error@RealData2ppm) >> reading colour map at line %d: r, g, b =", (i + 1));
            fprintf(stderr, " %3.3d %3.3d %3.3d\n", cm[0][i], cm[1][i], cm[2][i]);
            exit(1);
        }
    }
    fclose(ColMap);

    /* Write on unit 700 with PPM format */
    strcpy(fname, name);
    strcat(fname, ".ppm\0");

    ouni = fopen(fname, "w");
    if (!ouni) fprintf(stderr, "(Error@RealData2ppm) >> write access to file %s\n", fname);

    /*  Magic code */
    fprintf(ouni, "P3\n");

    /*  Dimensions */
    fprintf(ouni, "%d %d\n", s1, s2);

    /*  Maximum value */
    fprintf(ouni, "255\n");

    /*  Values from 0 to 255 */
    rmin = MinDoubleVal(s1 * s2, rdata); 
    rmax = MaxDoubleVal(s1 * s2, rdata);

    if ((*vmin == *vmax) && (*vmin == (double)0.0)) 
    {
        *vmin = rmin; 
        *vmax = rmax;
    }
    else 
    {
        rmin = *vmin; 
        rmax = *vmax;
    }

    vs = 0;
    for (i = 0; i < s1; i++) 
    {
        for (j = 0; j < s2; j++) 
        {
            vp = (int)((rdata[i + (j * s1)] - rmin) * 255.0 / (rmax - rmin));

            if (vp < 0) vp = 0;
            if (vp > 255) vp = 255;

            vs++;

            fprintf(ouni, " %3.3d %3.3d %3.3d", cm[0][vp], cm[1][vp], cm[2][vp]);

            if (vs >= 10) 
            {
                fprintf(ouni, " \n");
                vs = 0;
            }
        }
        fprintf(ouni, " ");
        vs = 0;
    }
    fclose(ouni);

    return;
}

void Statistics(int s1, int s2, double* rdata, int step)
{
    double mnv, mv, mxv, sd;
    int i, j;

    // OPP: Can mean value and standard deviation be computed together?

    // Compute MEAN VALUE 
    mv = 0.0;
    mnv = mxv = rdata[0];
    for (i = 0; i < s1; i++) 
    {
        for (j = 0; j < s2; j++) 
        {
            mv = mv + rdata[i + (j * s1)];
            if (mnv > rdata[i + (j * s1)]) mnv = rdata[i + (j * s1)];
            if (mxv < rdata[i + (j * s1)]) mxv = rdata[i + (j * s1)];
        }
    }
    mv = mv / (double)(s1 * s2);

    // Compute STANDARD DEVIATION
    sd = 0.0;
    for (i = 0; i < s1; i++) 
    {
        for (j = 0; j < s2; j++) 
        {
            sd = sd + (rdata[i + (j * s1)] - mv) * (rdata[i + (j * s1)] - mv);
        }
    }
    sd = sqrt(sd / (double)(s1 * s2));

    fprintf(stdout, ">> Step %4d: min, mean, max, std = %12.3e, %12.3e, %12.3e, %12.3e\n", step, mnv, mv, mxv, sd);

    return;
}

void Update(int xdots, int ydots, double* u1, double* u2)
{
    /* Compute next step using matrices g1, g2 of dimension (nr,nc) */

    int i, j;
    double CX, CY;
    double hx, dgx, hy, dgy, dd;

    dd = 0.0000001;
    hx = 1.0 / (double)xdots;
    hy = 1.0 / (double)ydots;
    dgx = -2.0 + hx * hx / (2 * dd);
    dgy = -2.0 + hy * hy / (2 * dd);
    CX = dd / (hx * hx);
    CY = dd / (hy * hy);

    for (j = 0; j < ydots - 1; j++) 
    {
        for (i = 0; i < xdots - 1; i++) 
        {
            if (i <= 0 || i >= xdots - 1) 
            {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            if (j <= 0 || j >= ydots - 1) 
            {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            u2[index2D(i, j, xdots)] = CX * (u1[index2D((i - 1), j, xdots)]
                                       + u1[index2D((i + 1), j, xdots)] + dgx * u1[index2D((i + 1), j, xdots)])
                                       + CY * (u1[index2D(i, (j - 1), xdots)]
                                       + u1[index2D(i, (j + 1), xdots)] + dgy * u1[index2D(i, j, xdots)]);
        }
    }

    for (j = 0; j < ydots - 1; j++) 
    {
        u2[index2D(0, j, xdots)] = u2[index2D(1, j, xdots)];
        u2[index2D(Xdots - 1, j, xdots)] = u2[index2D(Xdots - 2, j, xdots)];
    }

    for (i = 0; i < xdots - 1; i++) 
    {
        u2[index2D(i, 0, xdots)] = u2[index2D(i, 1, xdots)];
        u2[index2D(i, Ydots - 1, xdots)] = u2[index2D(i, Ydots - 2, xdots)];
    }

    return;
}