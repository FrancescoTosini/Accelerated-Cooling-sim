#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__ 
    #define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include "util.cu"

#define index2D(i,j,LD1) i + ((j)*LD1)    // element position in 2-D arrays
#define index3D(i,j,k,LD1,LD2) i + ((j)*LD1) + ((k)*LD1*LD2)   // element position in 3-D arrays

#define Xdots 1400   // Plate grid resolution in 2 dimensions
#define Ydots 1400   // May be changed to 1000x1000

// Parameters to compute point sensitiveness - values read from input file
double Sreal, Simag, Rreal, Rimag;
int MaxIters;
int TimeSteps;   // Evolution time steps

double* MeasuredValues;    // 2-D array - (NumInputValues,3) - Values read in input file
int NumInputValues;        // Number of values read in input file
double* TheorSlope;        // 2-D array - Theoretical value distribution
int TSlopeLength;          // TheorSlope grid dimensions
int* FieldWeight;          // 2-D array - (Xdots,Ydots) - Degree of sensitiveness to perturbing field 
double* FieldCoord;        // 3-D array - X, Y coordinates in field
double* FieldValues;       // 3-D array - X, Y coordinates in field

// Global GPU variables

__device__ int maxRow, maxCol;

/* CODE */

// Functions  prototypes

    void InitGrid(char* InputFile);
    int LinEquSolve(double* a, int n, double* b);
    void EqsDef(double x0, double x1, double y0, double y1, int N, int LA, double* A, double* Rhs, double* Pts);
    double Solution(double x, double y);
    void FieldDistribution();
    void GridDef(double x0, double x1, double y0, double y1, int N, double* Pts);
    void SensiblePoints(double Ir, double Ii, double Sr, double Si, int MaxIt);
    void FieldInit();
    double NearestValue(double xc, double yc, int ld, double* Values);
    void FieldPoints(double Diff);
    void Cooling(int steps);
    void RealData2ppm(int s1, int s2, double* rdata, double* vmin, double* vmax, char* name);
    void Statistics(int s1, int s2, double* rdata, int s);
    void Update(int xdots, int ydots, double* u1, double* u2);

// Accelerated functions

    void gpuInitGrid(char* InputFile);
    void gpuFieldDistribution();
    void gpuGridDef(double x0, double x1, double y0, double y1, int N, double* Pts);
    __global__ void gpuGridDefKernel(double x0, double y0, double dx, double dy, double* Pts, int Nm1, int len, int TSlopeLength);
    void gpuEqsDef(double x0, double x1, double y0, double y1, int N, int LA, double* A, double* Rhs, double* Pts);
    __global__ void gpuEqsDefKernel(double x0, double x1, double y0, double y1, int Nm1, double dx, double dy, int LA, double* A, double* Rhs, double* Pts, int TSlopeLength);
    __device__ double gpuSolution(double x, double y);

    int gpuLinEquSolve(double* a, int n, double* b);
    __global__ void gpuFindMax(double* maxima, int* maxIndex, double* a, int* ipiv, int n);
    __global__ void gpuGJStep(double* a, double* b, int n, int* indrow, int* indcol);

    __global__ void gpuLinEquSolveKernel(double* maxima, int* maxIndex, double* a, double* b, int* indrow, int* indcol, int* ipiv, int n);


int main(int argc, char* argv[])
{
    clock_t t0, t1, p0, p1;

    t0 = clock();
    printf(">> Starting\n");

    // Read input file
    p0 = clock();
    gpuInitGrid("Cooling.inp");
    p1 = clock();
    fprintf(stdout, ">> InitGrid ended in %lf seconds\n", (double)(p1 - p0)/CLOCKS_PER_SEC);

    // TheorSlope(TSlopeLength,3)
    p0 = clock();
    gpuFieldDistribution();
    p1 = clock();
    fprintf(stdout, ">> FieldDistribution ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    /*

    // FieldCoord(Xdots,Ydots,2), FieldWeight(Xdots,Ydots)
    p0 = clock();
    SensiblePoints(Sreal, Simag, Rreal, Rimag, MaxIters);
    p1 = clock();
    fprintf(stdout, ">> SensiblePoints ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    // MeasuredValues(:,3), FieldWeight(Xdots,Ydots) -> FieldValues(Xdots,Ydots,2)
    p0 = clock();
    FieldInit();
    p1 = clock();
    fprintf(stdout, ">> FieldInit ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    // FieldValues(Xdots,Ydots,2)
    p0 = clock();
    Cooling(TimeSteps);
    p1 = clock();
    fprintf(stdout, ">> Cooling ended in %lf seconds\n", (double)(p1 - p0) / CLOCKS_PER_SEC);

    t1 = clock();
    fprintf(stdout, ">> Computations ended in %lf seconds\n", (double)(t1 - t0) / CLOCKS_PER_SEC);

    */

    // End Program

    hipFree(MeasuredValues);
    hipFree(FieldWeight);
    hipFree(FieldCoord);
    hipFree(TheorSlope);
    free(FieldValues);

    return 0;

    /*time_t t0, t1, p0, p1;

    time(&t0);
    //fprintf(stdout, ">> Starting %s at: %s", argv[0], asctime(localtime(&t0)));
    printf(">> Starting\n");

    // Read input file
    time(&p0);
    InitGrid("Cooling.inp");
    time(&p1);
    fprintf(stdout, ">> InitGrid ended in %lf seconds\n", difftime(p1, p0));

    // TheorSlope(TSlopeLength,3)
    time(&p0);
    FieldDistribution();
    time(&p1);
    fprintf(stdout, ">> FieldDistribution ended in %lf seconds\n", difftime(p1, p0));

    // FieldCoord(Xdots,Ydots,2), FieldWeight(Xdots,Ydots)
    time(&p0);
    SensiblePoints(Sreal, Simag, Rreal, Rimag, MaxIters);
    time(&p1);
    fprintf(stdout, ">> SensiblePoints ended in %lf seconds\n", difftime(p1, p0));

    // MeasuredValues(:,3), FieldWeight(Xdots,Ydots) -> FieldValues(Xdots,Ydots,2)
    time(&p0);
    FieldInit();
    time(&p1);
    fprintf(stdout, ">> FieldInit ended in %lf seconds\n", difftime(p1, p0));

    // FieldValues(Xdots,Ydots,2)
    time(&p0);
    Cooling(TimeSteps);
    time(&p1);
    fprintf(stdout, ">> Cooling ended in %lf seconds\n", difftime(p1, p0));

    time(&t1);
    fprintf(stdout, ">> Ending at: %s", asctime(localtime(&t1)));
    fprintf(stdout, ">> Computations ended in %lf seconds\n", difftime(t1, t0));

    // End Program

    free(MeasuredValues);
    free(FieldWeight);
    free(FieldCoord);
    free(TheorSlope);
    free(FieldValues);

	return 0;*/
}

/* FUNCTIONS */

void InitGrid(char* InputFile)
{
    /* Output:
    !  MeasuredValues(:,3) - values read from input file
    !  Initialization of FieldWeight(Xdots,Ydots) and FieldCoord(Xdots,Ydots,2)
    */

    int valrows, st;
    char filerow[80];
    FILE* inpunit;

    fprintf(stdout, ">> Initializing grid ...\n");

    inpunit = fopen(InputFile, "r");
    if (!inpunit) 
    {
        fprintf(stderr, "(Error) >>> Cannot access file %s\n", InputFile);
        exit(-1);
    }

    // Read measured values
    NumInputValues = 0;
    valrows = 0;
    while (1)
    {
        st = readrow(filerow, 80, inpunit);
        if (filerow[0] == '#') continue;
        if (NumInputValues <= 0) 
        {
            if (sscanf(filerow, "  %d", &NumInputValues) < 1) 
            {
                if (NumInputValues <= 0) 
                {
                    fprintf(stderr, "(Error) >> there seems to be %d input values...\n", NumInputValues);
                    exit(-1);
                }
            }
            else 
            {
                MeasuredValues = (double*)malloc(sizeof(double) * NumInputValues * 3);
                if (MeasuredValues == NULL) 
                {
                    fprintf(stderr, "(Error) >> Cannot allocate MeasuredValues[%d,3] :(\n", NumInputValues);
                    exit(-1);
                }
            }
        }
        else 
        {
            if (sscanf(filerow, "%lf %lf %lf",
                &MeasuredValues[index2D(valrows, 0, NumInputValues)],  // X coord
                &MeasuredValues[index2D(valrows, 1, NumInputValues)],  // Y coord
                &MeasuredValues[index2D(valrows, 2, NumInputValues)])  // Measured value
                < 3) 
            {
                fprintf(stderr, "(Error) >>> something went wrong while reading MeasuredValues(%d,*)", valrows);
                exit(-1);
            }
            valrows++;
            if (valrows >= NumInputValues) break;
        }
    }

    /* Create and initialize FieldWeight */
    FieldWeight = (int*)malloc(sizeof(int) * Xdots * Ydots);
    if (FieldWeight == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldWeight[%d,%d]\n", Xdots, Ydots);
        exit(-1);
    }
    SetIntValue(FieldWeight, Xdots * Ydots, 0); // OPP: you can use calloc?

    /* Create and initialize FieldCoord */
    FieldCoord = (double*)malloc(sizeof(double) * Xdots * Ydots * 2);
    if (FieldCoord == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldCoord[%d,%d,2]\n", Xdots, Ydots);
        exit(-1);
    }
    SetDoubleValue(FieldCoord, Xdots * Ydots * 2, (double)0); // OPP: you can use calloc?

    /* Now read Sreal, Simag, Rreal, Rimag */
    Sreal = Simag = Rreal = Rimag = 0.0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Sreal from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#') continue;
        if (sscanf(filerow, "%lf", &Sreal) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Sreal from string.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Simag) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Simag from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rreal) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Rreal from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rimag) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read Rimag from input file.\n");
            exit(-1);
        }
        break;
    }

    /* Now read MaxIters */
    MaxIters = 0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1) continue;
        if (sscanf(filerow, "%d", &MaxIters) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from string.\n");
            exit(-1);
        }
        break;
    }

    /* Now read TimeSteps */
    TimeSteps = 0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1) continue;
        if (sscanf(filerow, "%d", &TimeSteps) < 1) 
        {
            fprintf(stderr, "(Error) >> Cannot read TimeSteps from string.\n");
            exit(-1);
        }
        break;
    }

    fclose(inpunit);
    return;
}

void gpuInitGrid(char* InputFile)
{
    /* Output:
    !  MeasuredValues(:,3) - values read from input file
    !  Initialization of FieldWeight(Xdots,Ydots) and FieldCoord(Xdots,Ydots,2)
    */

    int valrows, st;
    char filerow[80];
    FILE* inpunit;

    double* tmpMeasuredValues;
    hipError_t err;

    fprintf(stdout, "(NO CPU) >> Initializing grid ...\n");

    inpunit = fopen(InputFile, "r");
    if (!inpunit)
    {
        fprintf(stderr, "(Error) >>> Cannot access file %s\n", InputFile);
        exit(-1);
    }

    // Read measured values
    NumInputValues = 0;
    valrows = 0;
    while (1)
    {
        st = readrow(filerow, 80, inpunit);
        if (filerow[0] == '#') continue;
        if (NumInputValues <= 0)
        {
            if (sscanf(filerow, "  %d", &NumInputValues) < 1)
            {
                if (NumInputValues <= 0)
                {
                    fprintf(stderr, "(Error) >> there seems to be %d input values...\n", NumInputValues);
                    exit(-1);
                }
            }
            else
            {
                tmpMeasuredValues = (double*)malloc(sizeof(double) * NumInputValues * 3);
                if (tmpMeasuredValues == NULL)
                {
                    fprintf(stderr, "(Error) >> Cannot allocate tmpMeasuredValues[%d,3] :(\n", NumInputValues);
                    exit(-1);
                }
            }
        }
        else
        {
            if (sscanf(filerow, "%lf %lf %lf",
                &tmpMeasuredValues[index2D(valrows, 0, NumInputValues)],  // X coord
                &tmpMeasuredValues[index2D(valrows, 1, NumInputValues)],  // Y coord
                &tmpMeasuredValues[index2D(valrows, 2, NumInputValues)])  // Measured value
                < 3)
            {
                fprintf(stderr, "(Error) >>> something went wrong while reading MeasuredValues(%d,*)", valrows);
                exit(-1);
            }
            valrows++;
            if (valrows >= NumInputValues) break;
        }
    }

    err = hipMalloc(&MeasuredValues, sizeof(double) * NumInputValues * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldWeight[%d,%d] on GPU\n", Xdots, Ydots);
        exit(-1);
    }

    hipMemcpy(MeasuredValues, tmpMeasuredValues, sizeof(double) * NumInputValues * 3, hipMemcpyHostToDevice);
    free(tmpMeasuredValues);

    /* Create and initialize FieldWeight */
    err = hipMalloc(&FieldWeight, sizeof(int) * Xdots * Ydots);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldWeight[%d,%d] on GPU\n", Xdots, Ydots);
        exit(-1);
    }
    hipMemset(FieldWeight, 0, sizeof(int) * Xdots * Ydots);

    /* Create and initialize FieldCoord */
    err = hipMalloc(&FieldCoord, sizeof(double) * Xdots * Ydots * 2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error) >> Cannot allocate FieldCoord[%d,%d,2] on GPU\n", Xdots, Ydots);
        exit(-1);
    }
    hipMemset(FieldCoord, 0, sizeof(double) * Xdots * Ydots * 2);

    /* Now read Sreal, Simag, Rreal, Rimag */
    Sreal = Simag = Rreal = Rimag = 0.0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read Sreal from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#') continue;
        if (sscanf(filerow, "%lf", &Sreal) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read Sreal from string.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Simag) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read Simag from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rreal) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read Rreal from input file.\n");
            exit(-1);
        }
        if (fscanf(inpunit, "%lf", &Rimag) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read Rimag from input file.\n");
            exit(-1);
        }
        break;
    }

    /* Now read MaxIters */
    MaxIters = 0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1) continue;
        if (sscanf(filerow, "%d", &MaxIters) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from string.\n");
            exit(-1);
        }
        break;
    }

    /* Now read TimeSteps */
    TimeSteps = 0;
    while (1)
    {
        if (readrow(filerow, 80, inpunit) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read MaxIters from input file.\n");
            exit(-1);
        }
        if (filerow[0] == '#' || rowlen(filerow) < 1) continue;
        if (sscanf(filerow, "%d", &TimeSteps) < 1)
        {
            fprintf(stderr, "(Error) >> Cannot read TimeSteps from string.\n");
            exit(-1);
        }
        break;
    }

    fclose(inpunit);
    return;
}

void FieldDistribution()
{
    /*
    !  Compute theoretical value distribution of the perturbing field
    !  Output: TheorSlope(TSlopeLength,3) - theoretical field distribution function
    */
    double *CoeffMatrix, *B;
    double x0, y0, x1, y1;

    int M, Mm1, N, Nm1, LA;
    int i, rc;

    fprintf(stdout, "\t>> Computing theoretical perturbing field...\n");

    x0 = Sreal; 
    y0 = Simag; 
    x1 = x0 + Rreal; 
    y1 = y0 + Rimag;

    // How many intervals? It should be safe to use SQRT(Xdots)
    M = sqrt((double)Xdots);
    N = sqrt((double)Ydots);

    Nm1 = N - 1;  // Grid points minus boundary
    Mm1 = M - 1;  // Grid points minus boundary

    LA = Mm1 * Nm1; // unknown points
    TSlopeLength = LA;

    /* Allocate CoeffMatrix */
    CoeffMatrix = (double*)malloc(sizeof(double) * LA * LA);
    if (CoeffMatrix == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate CoeffMatrix[%d,%d]\n", LA, LA);
        exit(-1);
    }

    /* Allocate TheorSlope */
    TheorSlope = (double*)malloc(sizeof(double) * TSlopeLength * 3);
    if (TheorSlope == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate TheorSlope[%d,3]\n", TSlopeLength);
        exit(-1);
    }

    /* Allocate B */
    B = (double*)malloc(sizeof(double) * LA);
    if (B == NULL) 
    {
        fprintf(stderr, "(Error) >> Cannot allocate B[%d]\n", LA);
        exit(-1);
    }

    GridDef(x0, x1, y0, y1, N, TheorSlope);

    EqsDef(x0, x1, y0, y1, N, LA, CoeffMatrix, B, TheorSlope);

    rc = LinEquSolve(CoeffMatrix, LA, B);
    if (rc != 0) exit(-1);

    for (i = 0; i < LA; i++) TheorSlope[index2D(i, 2, TSlopeLength)] = B[i]; // OPP: why not use memcpy?

    free(CoeffMatrix);
    free(B);

    return;
}

void gpuFieldDistribution()
{
    /*
    !  Compute theoretical value distribution of the perturbing field
    !  Output: TheorSlope(TSlopeLength,3) - theoretical field distribution function
    */
    double* CoeffMatrix, * B;
    double x0, y0, x1, y1;

    int M, Mm1, N, Nm1, LA;
    int i, rc;

    hipError_t err;

    fprintf(stdout, "\t>> Computing theoretical perturbing field...\n");

    x0 = Sreal;
    y0 = Simag;
    x1 = x0 + Rreal;
    y1 = y0 + Rimag;

    // How many intervals? It should be safe to use SQRT(Xdots)
    M = sqrt((double)Xdots);
    N = sqrt((double)Ydots);

    Nm1 = N - 1;  // Grid points minus boundary
    Mm1 = M - 1;  // Grid points minus boundary

    LA = Mm1 * Nm1; // unknown points
    TSlopeLength = LA;

    /* Allocate CoeffMatrix */
    err = hipMalloc(&CoeffMatrix, sizeof(double) * LA * LA);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error) >> Cannot allocate CoeffMatrix[%d,%d] on GPU\n", LA, LA);
        exit(-1);
    }

    /* Allocate TheorSlope */
    err = hipMalloc(&TheorSlope, sizeof(double) * TSlopeLength * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error) >> Cannot allocate TheorSlope[%d,3] on GPU\n", TSlopeLength);
        exit(-1);
    }

    /* Allocate B */
    err = hipMalloc(&B, sizeof(double) * LA);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error) >> Cannot allocate B[%d] on GPU\n", LA);
        exit(-1);
    }

    gpuGridDef(x0, x1, y0, y1, N, TheorSlope);
    hipDeviceSynchronize();

    gpuEqsDef(x0, x1, y0, y1, N, LA, CoeffMatrix, B, TheorSlope);
    hipDeviceSynchronize();

    gpuLinEquSolve(CoeffMatrix, LA, B);
    hipDeviceSynchronize();

    //if (rc != 0) exit(-1);

    /*
    for (i = 0; i < LA; i++) TheorSlope[index2D(i, 2, TSlopeLength)] = B[i]; // OPP: why not use memcpy?
    */

    hipFree(CoeffMatrix);
    hipFree(B);

    return;
}

void SensiblePoints(double Ir, double Ii, double Sr, double Si, int MaxIt)
{
    /*
    !  Compute "heated" points
    !  Output:
    !          FieldCoord(Xdots,Ydots,2)
    !          FieldWeight(Xdots,Ydots)
   */

    int ix, iy, iz;
    double ca, cb, za, zb;
    double rad, zan, zbn;
    double Xinc, Yinc;

    fprintf(stdout, "\t>> Computing sensitivity to field effects...\n");

    Xinc = Sr / (double)Xdots;
    Yinc = Si / (double)Ydots;

    for (iy = 0; iy < Ydots; iy++) 
    {
        for (ix = 0; ix < Xdots; ix++) 
        {
            ca = Xinc * ix + Ir;
            cb = Yinc * iy + Ii;
            FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)] = ca;
            FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)] = cb;
            rad = ca * ca * ((double)1.0 + (cb / ca) * (cb / ca));
            zan = 0.0;
            zbn = 0.0;
            for (iz = 1; iz <= MaxIt; iz++) 
            {
                if (rad > (double)4.0) break;
                za = zan;
                zb = zbn;
                zan = ca + (za - zb) * (za + zb);
                zbn = 2.0 * (za * zb + cb / 2.0);
                rad = zan * zan * ((double)1.0 + (zbn / zan) * (zbn / zan));
            }
            FieldWeight[index2D(ix, iy, Xdots)] = iz;
        }
    }

    return;
}

void FieldInit()
{
    /*
    ! Initialize field values in the grid. Values are computed on the basis
    ! of the measured values read in subroutine InitGrid and the gross grid
    ! values computed in subroutine FieldDistribution. Moreover sensitiveness
    ! to field effects as computed in subroutine SensiblePoints are taken into
    ! account.
    !
    ! Input:
    !        MeasuredValues(:,3)
    !        FieldWeight(Xdots,Ydots)
    ! Output:
    !        FieldValues(Xdots,Ydots,2)
    */

    int rv;
    double xc, yc, ev, sv, sd, DiscrValue;
    double *DiffValues;

    fprintf(stdout, "\t>> Initializing entity of field effects...\n");

    /* Allocate FieldValues */
    FieldValues = (double*)malloc(sizeof(double) * Xdots * Ydots * 2);
    if (FieldValues == NULL) 
    {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate FieldValues[%d,%d,2]\n", Xdots, Ydots);
        exit(-1);
    }
    SetDoubleValue(FieldValues, Xdots * Ydots * 2, (double)0); // OPP: you can use calloc?

    /* Allocate DiffValues */
    DiffValues = (double*)malloc(sizeof(double) * NumInputValues);
    if (DiffValues == NULL) 
    {
        fprintf(stderr, "(Error@FieldInit) >> Cannot allocate DiffValues[%d]\n", NumInputValues);
        exit(-1);
    }
    SetDoubleValue(DiffValues, NumInputValues, (double)0.0); // OPP: you can use calloc?

    /* Compute discrepancy between Measured and Theoretical value */
    DiscrValue = 0.0;
    for (rv = 0; rv < NumInputValues; rv++) 
    {
        xc = MeasuredValues[index2D(rv, 0, NumInputValues)];
        yc = MeasuredValues[index2D(rv, 1, NumInputValues)];

        // TheorSlope is computed on the basis of a coarser grid, so look for the best values near xc, yc coordinates
        sv = NearestValue(xc, yc, TSlopeLength, TheorSlope);
        ev = MeasuredValues[index2D(rv, 2, NumInputValues)];

        DiffValues[rv] = ev - sv;
        DiscrValue += ev - sv;
    }
    DiscrValue = DiscrValue / (double)NumInputValues;

    // Compute standard deviation
    sd = 0.0;
    for (rv = 0; rv < NumInputValues; rv++) sd = sd + (DiffValues[rv] - DiscrValue) * (DiffValues[rv] - DiscrValue);
    sd = sqrt(sd / (double)NumInputValues);

    // Print statistics
    fprintf(stdout, "\t...Number of Points, Mean value, Standard deviation = %d, %12.3e, %12.3e\n", NumInputValues, DiscrValue, sd);

    // Compute FieldValues stage 1
    FieldPoints(DiscrValue);

    free(DiffValues);

    return;
}

void Cooling(int steps)
{
    /*
    !  Compute evolution of the effects of the field
    !  Input/Output:
    !                FieldValues(Xdots,Ydots,2)
    */

    int iz, it;
    char fname[80];
    double vmin, vmax;

    fprintf(stdout, "\t>> Computing cooling of field effects ...\n");
    fprintf(stdout, "\t... %d steps ...\n", steps);
    sprintf(fname, "FieldValues0000");

    vmin = vmax = 0.0;
    //RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], &vmin, &vmax, fname);
    Statistics(Xdots, Ydots, &FieldValues[index3D(0, 0, 0, Xdots, Ydots)], 0);

    iz = 1;
    for (it = 1; it <= steps; it++) 
    {
        // Update the value of grid points
        Update(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &FieldValues[index3D(0, 0, 2 - iz, Xdots, Ydots)]);
        iz = 3 - iz;

        // Print and show results 
        sprintf(fname, "FieldValues%4.4d", it);
        //if (it % 4 == 0) RealData2ppm(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], &vmin, &vmax, fname);
        Statistics(Xdots, Ydots, &FieldValues[index3D(0, 0, iz - 1, Xdots, Ydots)], it);
    }

    return;
}

/* SUB-FUNCTIONS */

void GridDef(double x0, double x1, double y0, double y1, int N, double* Pts)
{
    double x, y, dx, dy;
    int i, j, np, Mm1, Nm1;

    Mm1 = sqrt((double)Xdots) - 1;
    Nm1 = sqrt((double)Ydots) - 1;
    dx = (x1 - x0) / (double)N; 
    dy = (y1 - y0) / (double)N;

    np = -1;
    for (i = 0; i < Mm1; i++) 
    {
        for (j = 0; j < Nm1; j++) 
        {
            np++;
            if (np > Mm1 * Nm1) 
            {
                fprintf(stderr, "(Error@GridDef) >> NP = %d > N*N = %d\n", np, Nm1 * Nm1);
                exit(-1);
            }
            x = x0 + dx * (double)(i + 1);
            y = y0 + dy * (double)(j + 1);
            Pts[index2D(np, 0, TSlopeLength)] = x;
            Pts[index2D(np, 1, TSlopeLength)] = y;
        }
    }
    return;
}

void gpuGridDef(double x0, double x1, double y0, double y1, int N, double* Pts)
{
    double dx, dy;
    int Mm1, Nm1;

    Mm1 = sqrt((double)Xdots) - 1;
    Nm1 = sqrt((double)Ydots) - 1;
    dx = (x1 - x0) / (double)N;
    dy = (y1 - y0) / (double)N;

    gpuGridDefKernel<<<6, 128>>>(x0, y0, dx, dy, Pts, Nm1, Nm1 * Mm1, TSlopeLength);

    return;
}

__global__
void gpuGridDefKernel(double x0, double y0, double dx, double dy, double* Pts, int Nm1, int len, int TSlopeLength)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    int np, i, j;
    double x, y;

    for (np = id; np < len; np += gridSize)
    {
        i = np / Nm1;
        j = np % Nm1;

        x = x0 + dx * (double)(i + 1);
        y = y0 + dy * (double)(j + 1);
        Pts[index2D(i, 0, TSlopeLength)] = x;
        Pts[index2D(i, 1, TSlopeLength)] = y;
    }
}

void EqsDef(double x0, double x1, double y0, double y1, int N, int LA, double* A, double* Rhs, double* Pts)
{
    // Pts(LA,3) - inner grid point Coordinates
    // Rhs(LA)   - Linear equation Right Hand Side
    // A(LA,LA)  - Linear equation matrix

    double x, y, Eps, dx, dy;
    int np, Nm1, pos;

    //  Define A matrix and RHS

    Nm1 = N - 1;
    dx = (x1 - x0) / (double)N; 
    dy = (y1 - y0) / (double)N;

    SetDoubleValue(A, LA * LA, (double)0); // OPP: you can use calloc?
    SetDoubleValue(Rhs, LA, (double)0); // OPP: you can use calloc?

    for (np = 0; np < LA; np++) 
    {
        x = Pts[index2D(np, 0, TSlopeLength)];
        y = Pts[index2D(np, 1, TSlopeLength)];

        A[index2D(np, np, LA)] = -4.0;

        Rhs[np] = (x + y) * dx * dy;

        // define Eps function of grid dimensions 
        Eps = (dx + dy) / 20.0;

        // where is P(x-dx,y) ? 
        if (fabs((x - dx) - x0) < Eps) Rhs[np] = Rhs[np] - Solution(x0, y);
        else 
        {
            // Find pos = position of P(x-dx,y)
            pos = np - Nm1;
            if (fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x - dx)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> x-dx: pos, np, d = %d %d %lf\n", pos, np, fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x - dx)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x+dx,y) ? 
        if (fabs((x + dx) - x1) < Eps) Rhs[np] = Rhs[np] - Solution(x1, y);
        else 
        {
            // Find pos = position of P(x+dx,y)
            pos = np + Nm1;
            if (fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x + dx)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> x+dx: %lf\n", fabs(Pts[index2D(pos, 0, TSlopeLength)] - (x + dx)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y-dy) ? 
        if (fabs((y - dy) - y0) < Eps) Rhs[np] = Rhs[np] - Solution(x, y0);
        else 
        {
            // Find pos = position of P(x,y-dy)
            pos = np - 1;
            if (fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y - dy)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> y-dy: %lf\n", fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y - dy)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y+dy) ? 
        if (fabs((y + dy) - y1) < Eps) Rhs[np] = Rhs[np] - Solution(x, y1);
        else 
        {
            // Find pos = position of P(x,y-dy)
            pos = np + 1;
            if (fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y + dy)) > Eps) 
            {
                fprintf(stderr, "(Error@EqsDef) >> y+dy: %lf\n", fabs(Pts[index2D(pos, 1, TSlopeLength)] - (y + dy)));
                exit(-1);
            }
            A[index2D(np, pos, LA)] = 1.0;
        }
    }
    return;
}

void gpuEqsDef(double x0, double x1, double y0, double y1, int N, int LA, double* A, double* Rhs, double* Pts)
{
    // Pts(LA,3) - inner grid point Coordinates
    // Rhs(LA)   - Linear equation Right Hand Side
    // A(LA,LA)  - Linear equation matrix

    double x, y, Eps, dx, dy;
    int np, Nm1, pos;

    //  Define A matrix and RHS

    Nm1 = N - 1;
    dx = (x1 - x0) / (double)N;
    dy = (y1 - y0) / (double)N;

    hipMemset(A, 0, sizeof(double) * LA * LA);
    hipMemset(Rhs, 0, sizeof(double) * LA);

    gpuEqsDefKernel<<<6, 128>>>(x0, x1, y0, y1, Nm1, dx, dy, LA, A, Rhs, Pts, TSlopeLength);
    
    return;
}

__global__
void gpuEqsDefKernel(double x0, double x1, double y0, double y1, int Nm1, double dx, double dy, int LA, double* A, double* Rhs, double* Pts, int TSlopeLength)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    int np, pos;
    double x, y, Eps;

    for (np = id; np < LA; np += gridSize)
    {
        x = Pts[index2D(np, 0, TSlopeLength)];
        y = Pts[index2D(np, 1, TSlopeLength)];

        A[index2D(np, np, LA)] = -4.0;

        Rhs[np] = (x + y) * dx * dy;

        // define Eps function of grid dimensions 
        Eps = (dx + dy) / 20.0;

        // where is P(x-dx,y) ? 
        if (fabs((x - dx) - x0) < Eps) Rhs[np] = Rhs[np] - gpuSolution(x0, y);
        else
        {
            // Find pos = position of P(x-dx,y)
            pos = np - Nm1;
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x+dx,y) ? 
        if (fabs((x + dx) - x1) < Eps) Rhs[np] = Rhs[np] - gpuSolution(x1, y);
        else
        {
            // Find pos = position of P(x+dx,y)
            pos = np + Nm1;
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y-dy) ? 
        if (fabs((y - dy) - y0) < Eps) Rhs[np] = Rhs[np] - gpuSolution(x, y0);
        else
        {
            // Find pos = position of P(x,y-dy)
            pos = np - 1;
            A[index2D(np, pos, LA)] = 1.0;
        }

        // where is P(x,y+dy) ? 
        if (fabs((y + dy) - y1) < Eps) Rhs[np] = Rhs[np] - gpuSolution(x, y1);
        else
        {
            // Find pos = position of P(x,y-dy)
            pos = np + 1;
            A[index2D(np, pos, LA)] = 1.0;
        }
    }
}

double Solution(double x, double y)
{
    return ((x * x * x) + (y * y * y)) / (double)6.0;
}

__device__
double gpuSolution(double x, double y)
{
    return ((x * x * x) + (y * y * y)) / (double)6.0;
}

int LinEquSolve(double* a, int n, double* b)
{
    /* Gauss-Jordan elimination algorithm */
    int i, j, k, l, icol, irow;
    int *indcol, *indrow, *ipiv;
    double bigger, temp;

    /* Allocate indcol */
    indcol = (int*)malloc(sizeof(int) * n);
    if (indcol == NULL) 
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indcol[%d]\n", n);
        return(-1);
    }

    /* Allocate indrow */
    indrow = (int*)malloc(sizeof((int)1) * n);
    if (indrow == NULL) 
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indrow[%d]\n", n);
        return(-1);
    }

    /* Allocate ipiv */
    ipiv = (int*)malloc(sizeof((int)1) * n);
    if (ipiv == NULL) 
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate ipiv[%d]\n", n);
        return(-1);
    }
    SetIntValue(ipiv, n, 0); // OPP: you can use calloc?

    /* Actual algorithm */

    for (i = 0; i < n; i++) 
    {
        bigger = 0.0;

        for (j = 0; j < n; j++) 
        {
            if (ipiv[j] != 1) 
            {
                for (k = 0; k < n; k++) 
                {
                    if (ipiv[k] == 0 && bigger <= fabs(a[index2D(j, k, n)])) 
                    {
                        bigger = fabs(a[index2D(j, k, n)]);
                        irow = j;
                        icol = k;
                    }
                }
            }
        }

        ipiv[icol] = ipiv[icol] + 1;

        if (irow != icol) 
        {
            for (l = 0; l < n; l++) 
            {
                temp = a[index2D(irow, l, n)];
                a[index2D(irow, l, n)] = a[index2D(icol, l, n)];
                a[index2D(icol, l, n)] = temp;
            }
            temp = b[irow];
            b[irow] = b[icol];
            b[icol] = temp;
        }

        indrow[i] = irow;
        indcol[i] = icol;

        if (a[index2D(icol, icol, n)] == 0.0) 
        {
            fprintf(stderr, "(Error@LinEquSolve) >> a(%d,%d): singular matrix!", icol, icol);
            return -2;
        }

        temp = (double)1.0 / a[index2D(icol, icol, n)];
        a[index2D(icol, icol, n)] = 1.0;

        for (l = 0; l < n; l++) a[index2D(icol, l, n)] = a[index2D(icol, l, n)] * temp;

        b[icol] = b[icol] * temp;

        for (l = 0; l < n; l++) 
        {
            if (l != icol) 
            {
                temp = a[index2D(l, icol, n)];
                a[index2D(l, icol, n)] = 0.0;
                for (k = 0; k < n; k++) 
                {
                    a[index2D(l, k, n)] = a[index2D(l, k, n)] - a[index2D(icol, k, n)] * temp;
                }
                b[l] = b[l] - b[icol] * temp;
            }
        }
    }

    for (l = n - 1; l >= 0; l--) 
    {
        if (indrow[l] != indcol[l]) 
        {
            for (k = 0; k < n; k++) 
            {
                temp = a[index2D(k, indrow[l], n)];
                a[index2D(k, indrow[l], n)] = a[index2D(k, indcol[l], n)];
                a[index2D(k, indcol[l], n)] = temp;
            }
        }
    }

    free(indcol);
    free(indrow);
    free(ipiv);

    return 0;
}

int gpuLinEquSolve(double* a, int n, double* b)
{
    /* Gauss-Jordan elimination algorithm */
    int *indcol, *indrow, *ipiv;

    hipError_t err;

    /* Allocate indcol */
    err = hipMalloc(&indcol, sizeof(int) * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indcol[%d] on GPU\n", n);
        return(-1);
    }

    /* Allocate indrow */
    err = hipMalloc(&indrow, sizeof(int) * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate indrow[%d] on GPU\n", n);
        return(-1);
    }

    /* Allocate ipiv */
    err = hipMalloc(&ipiv, sizeof(int) * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate ipiv[%d] on GPU\n", n);
        return(-1);
    }
    hipMemset(ipiv, 0, sizeof(int) * n);

    /* Actual algorithm */

    int* maxIndex;
    double* maxima;

    err = hipMalloc(&maxIndex, sizeof(int) * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate maxIndex on GPU\n", n);
        return(-1);
    }

    err = hipMalloc(&maxima, sizeof(double) * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(Error@LinEquSolve) >> Cannot allocate maxima on GPU\n", n);
        return(-1);
    }

    gpuLinEquSolveKernel<<<1, 1024>>>(maxima, maxIndex, a, b, indrow, indcol, ipiv, n);
    hipDeviceSynchronize();

    hipFree(indcol);
    hipFree(indrow);
    hipFree(ipiv);
    hipFree(maxIndex);
    hipFree(maxima);

    return 0;
}

__global__
void gpuFindMax(double* maxima, int* maxIndex, double* a, int* ipiv, int n)
{
    int j, k, icol, redSize;
    double max;

    for (j = threadIdx.x; j < n; j += blockDim.x)
    {
        max = 0;

        if (ipiv[j] != 1)
        {
            for (k = 0; k < n; k++)
            {
                if (ipiv[k] == 0 && max <= fabs(a[index2D(j, k, n)]))
                {
                    max = fabs(a[index2D(j, k, n)]);
                    icol = k;
                }
            }
        }

        maxima[j] = max;
        maxIndex[j] = icol;
    }

    __syncthreads();

    for (j = threadIdx.x; j < n; j += blockDim.x)
    {
        if (j % 2 == 0 && (j+1) < n)
        {
            if (maxima[j] < maxima[j + 1])
            {
                maxima[j] = maxima[j + 1];
                maxIndex[j] = maxIndex[j + 1];
            }
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        icol = 0;

        for (j = 2; j < n; j += 2)
            if (maxima[j] > maxima[icol]) icol = j;

        maxRow = icol;
        maxCol = maxIndex[icol];

        ipiv[maxCol] = ipiv[maxCol] + 1;
    }
}

__global__
void gpuGJStep(double* a, double* b, int n, int* indrow, int* indcol)
{
    int i, k;
    double tmp;

    __shared__ double temp;

    if (maxRow != maxCol) 
    {
        for (i = threadIdx.x; i < n; i += blockDim.x)
        {
            tmp = a[index2D(maxRow, i, n)];
            a[index2D(maxRow, i, n)] = a[index2D(maxCol, i, n)];
            a[index2D(maxCol, i, n)] = tmp;
        }

        if (threadIdx.x == 0)
        {
            tmp = b[maxRow];
            b[maxRow] = b[maxCol];
            b[maxCol] = tmp;
        }
    }

    if (threadIdx.x == 0)
    {
        indrow[i] = maxRow;
        indcol[i] = maxCol;
    }

    __syncthreads();

    // TODO: Missing check on singularity

    if (threadIdx.x == 0)
    {
        temp = (double)1.0 / a[index2D(maxCol, maxCol, n)];

        //TODO: check if it is an error
        //a[index2D(maxCol, maxCol, n)] = 1.0; 

        b[maxCol] = b[maxCol] * temp;
    }

    __syncthreads();

    for (i = threadIdx.x; i < n; i += blockDim.x)
        a[index2D(maxCol, i, n)] = a[index2D(maxCol, i, n)] * temp;

    __syncthreads();

    for (i = threadIdx.x; i < n; i += blockDim.x)
    {
        if (i != maxCol)
        {
            tmp = a[index2D(i, maxCol, n)];
            a[index2D(i, maxCol, n)] = 0.0;
            for (k = 0; k < n; k++)
                a[index2D(i, k, n)] = a[index2D(i, k, n)] - a[index2D(maxCol, k, n)] * tmp;
            b[i] = b[i] - b[maxCol] * tmp;
        }
    }
}

__global__
void gpuLinEquSolveKernel(double* maxima, int* maxIndex, double* a, double* b, int* indrow, int* indcol, int* ipiv, int n)
{
    int iter, i, j, k, icol, redSize;
    double max, tmp;

    __shared__ double temp;

    for (iter = 0; iter < n; iter++)
    {
        for (j = threadIdx.x; j < n; j += blockDim.x)
        {
            max = 0;

            if (ipiv[j] != 1)
            {
                for (k = 0; k < n; k++)
                {
                    if (ipiv[k] == 0 && max <= fabs(a[index2D(j, k, n)]))
                    {
                        max = fabs(a[index2D(j, k, n)]);
                        icol = k;
                    }
                }
            }

            maxima[j] = max;
            maxIndex[j] = icol;
        }

        __syncthreads();

        for (j = threadIdx.x; j < n; j += blockDim.x)
        {
            if (j % 2 == 0 && (j + 1) < n)
            {
                if (maxima[j] < maxima[j + 1])
                {
                    maxima[j] = maxima[j + 1];
                    maxIndex[j] = maxIndex[j + 1];
                }
            }
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            icol = 0;

            for (j = 2; j < n; j += 2)
                if (maxima[j] > maxima[icol]) icol = j;

            maxRow = icol;
            maxCol = maxIndex[icol];

            ipiv[maxCol] = ipiv[maxCol] + 1;
        }

        __syncthreads();

        if (maxRow != maxCol)
        {
            for (i = threadIdx.x; i < n; i += blockDim.x)
            {
                tmp = a[index2D(maxRow, i, n)];
                a[index2D(maxRow, i, n)] = a[index2D(maxCol, i, n)];
                a[index2D(maxCol, i, n)] = tmp;
            }

            if (threadIdx.x == 0)
            {
                tmp = b[maxRow];
                b[maxRow] = b[maxCol];
                b[maxCol] = tmp;
            }
        }

        if (threadIdx.x == 0)
        {
            indrow[i] = maxRow;
            indcol[i] = maxCol;
        }

        __syncthreads();

        // TODO: Missing check on singularity

        if (threadIdx.x == 0)
        {
            temp = (double)1.0 / a[index2D(maxCol, maxCol, n)];

            //TODO: check if it is an error
            //a[index2D(maxCol, maxCol, n)] = 1.0; 

            b[maxCol] = b[maxCol] * temp;
        }

        __syncthreads();

        for (i = threadIdx.x; i < n; i += blockDim.x)
            a[index2D(maxCol, i, n)] = a[index2D(maxCol, i, n)] * temp;

        __syncthreads();

        for (i = threadIdx.x; i < n; i += blockDim.x)
        {
            if (i != maxCol)
            {
                tmp = a[index2D(i, maxCol, n)];
                a[index2D(i, maxCol, n)] = 0.0;
                for (k = 0; k < n; k++)
                    a[index2D(i, k, n)] = a[index2D(i, k, n)] - a[index2D(maxCol, k, n)] * tmp;
                b[i] = b[i] - b[maxCol] * tmp;
            }
        }

        __syncthreads();
    }

    // Last step

    for (j = threadIdx.x; j < n; j += blockDim.x)
    {
        if (indrow[j] != indcol[j])
        {
            for (k = 0; k < n; k++)
            {
                tmp = a[index2D(k, indrow[j], n)];
                a[index2D(k, indrow[j], n)] = a[index2D(k, indcol[j], n)];
                a[index2D(k, indcol[j], n)] = tmp;
            }
        }
    }
}

double NearestValue(double xc, double yc, int ld, double* Values)
{
    // look for the best values near xc, yc coordinates
    double v;

    double d, md; // minimum distance
    int np; // number of nearest points
    int i;

    md = ((xc - Values[index2D(0, 0, ld)]) * (xc - Values[index2D(0, 0, ld)])) +
         ((yc - Values[index2D(0, 1, ld)]) * (yc - Values[index2D(0, 1, ld)]));

    // Compute lowest distance
    for (i = 0; i < ld; i++) 
    {
        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));
        if (md > d) md = d;
    }

    np = 0;
    v = 0.0;

    // Compute nearest value
    for (i = 0; i < ld; i++) 
    {
        d = ((xc - Values[index2D(i, 0, ld)]) * (xc - Values[index2D(i, 0, ld)])) +
            ((yc - Values[index2D(i, 1, ld)]) * (yc - Values[index2D(i, 1, ld)]));
        if (md == d) 
        {
            // add contributed value
            np = np + 1;
            v = v + Values[index2D(i, 2, ld)];
        }
    }

    // mean value
    v = v / (double)np;

    return v;
}

void FieldPoints(double Diff)
{
    int ix, iy;
    double xc, yc, sv;
    double rmin, rmax;

    rmax = MaxIntVal(Xdots * Ydots, FieldWeight);
    rmin = MinIntVal(Xdots * Ydots, FieldWeight);

    for (iy = 0; iy < Ydots; iy++) 
    {
        for (ix = 0; ix < Xdots; ix++) 
        {
            xc = FieldCoord[index3D(ix, iy, 0, Xdots, Ydots)];
            yc = FieldCoord[index3D(ix, iy, 1, Xdots, Ydots)];

            // Compute effects of field in every point
            sv = NearestValue(xc, yc, TSlopeLength, TheorSlope);
            FieldValues[index3D(ix, iy, 0, Xdots, Ydots)] = 293.16 + 80 * (Diff + sv) * (FieldWeight[index2D(ix, iy, Xdots)] - rmin) / (rmax - rmin);
        }
    }

    // Copy initial status 
    // OPP: use memcpy?
    for (iy = 0; iy < Ydots; iy++) 
    {
        for (ix = 0; ix < Xdots; ix++) 
        {
            FieldValues[index3D(ix, iy, 1, Xdots, Ydots)] = FieldValues[index3D(ix, iy, 0, Xdots, Ydots)];
        }
    }

    return;
}

void RealData2ppm(int s1, int s2, double* rdata, double* vmin, double* vmax, char* name)
{
    /* Simple subroutine to dump integer data in a PPM format */

    int cm[3][256];  /* R,G,B, Colour Map */
    FILE* ouni, * ColMap;
    int i, j, rc, vp, vs;
    double  rmin, rmax;
    char  fname[80], jname[80], command[80];

    /* Load color map: 256 colours */
    ColMap = fopen("ColorMap.txt", "r");
    if (ColMap == NULL) 
    {
        fprintf(stderr, "(Error@RealData2ppm) >> Cannot open ColorMap.txt\n");
        exit(-1);
    }
    for (i = 0; i < 256; i++) 
    {
        if (fscanf(ColMap, " %3d %3d %3d", &cm[0][i], &cm[1][i], &cm[2][i]) < 3) 
        {
            fprintf(stderr, "(Error@RealData2ppm) >> reading colour map at line %d: r, g, b =", (i + 1));
            fprintf(stderr, " %3.3d %3.3d %3.3d\n", cm[0][i], cm[1][i], cm[2][i]);
            exit(1);
        }
    }
    fclose(ColMap);

    /* Write on unit 700 with PPM format */
    strcpy(fname, name);
    strcat(fname, ".ppm\0");

    ouni = fopen(fname, "w");
    if (!ouni) fprintf(stderr, "(Error@RealData2ppm) >> write access to file %s\n", fname);

    /*  Magic code */
    fprintf(ouni, "P3\n");

    /*  Dimensions */
    fprintf(ouni, "%d %d\n", s1, s2);

    /*  Maximum value */
    fprintf(ouni, "255\n");

    /*  Values from 0 to 255 */
    rmin = MinDoubleVal(s1 * s2, rdata); 
    rmax = MaxDoubleVal(s1 * s2, rdata);

    if ((*vmin == *vmax) && (*vmin == (double)0.0)) 
    {
        *vmin = rmin; 
        *vmax = rmax;
    }
    else 
    {
        rmin = *vmin; 
        rmax = *vmax;
    }

    vs = 0;
    for (i = 0; i < s1; i++) 
    {
        for (j = 0; j < s2; j++) 
        {
            vp = (int)((rdata[i + (j * s1)] - rmin) * 255.0 / (rmax - rmin));

            if (vp < 0) vp = 0;
            if (vp > 255) vp = 255;

            vs++;

            fprintf(ouni, " %3.3d %3.3d %3.3d", cm[0][vp], cm[1][vp], cm[2][vp]);

            if (vs >= 10) 
            {
                fprintf(ouni, " \n");
                vs = 0;
            }
        }
        fprintf(ouni, " ");
        vs = 0;
    }
    fclose(ouni);

    return;
}

void Statistics(int s1, int s2, double* rdata, int step)
{
    double mnv, mv, mxv, sd;
    int i, j;

    // OPP: Can mean value and standard deviation be computed together?

    // Compute MEAN VALUE 
    mv = 0.0;
    mnv = mxv = rdata[0];
    for (i = 0; i < s1; i++) 
    {
        for (j = 0; j < s2; j++) 
        {
            mv = mv + rdata[i + (j * s1)];
            if (mnv > rdata[i + (j * s1)]) mnv = rdata[i + (j * s1)];
            if (mxv < rdata[i + (j * s1)]) mxv = rdata[i + (j * s1)];
        }
    }
    mv = mv / (double)(s1 * s2);

    // Compute STANDARD DEVIATION
    sd = 0.0;
    for (i = 0; i < s1; i++) 
    {
        for (j = 0; j < s2; j++) 
        {
            sd = sd + (rdata[i + (j * s1)] - mv) * (rdata[i + (j * s1)] - mv);
        }
    }
    sd = sqrt(sd / (double)(s1 * s2));

    fprintf(stdout, ">> Step %4d: min, mean, max, std = %12.3e, %12.3e, %12.3e, %12.3e\n", step, mnv, mv, mxv, sd);

    return;
}

void Update(int xdots, int ydots, double* u1, double* u2)
{
    /* Compute next step using matrices g1, g2 of dimension (nr,nc) */

    int i, j;
    double CX, CY;
    double hx, dgx, hy, dgy, dd;

    dd = 0.0000001;
    hx = 1.0 / (double)xdots;
    hy = 1.0 / (double)ydots;
    dgx = -2.0 + hx * hx / (2 * dd);
    dgy = -2.0 + hy * hy / (2 * dd);
    CX = dd / (hx * hx);
    CY = dd / (hy * hy);

    for (j = 0; j < ydots - 1; j++) 
    {
        for (i = 0; i < xdots - 1; i++) 
        {
            if (i <= 0 || i >= xdots - 1) 
            {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            if (j <= 0 || j >= ydots - 1) 
            {
                u2[index2D(i, j, xdots)] = u1[index2D(i, j, xdots)];
                continue;
            }

            u2[index2D(i, j, xdots)] = CX * (u1[index2D((i - 1), j, xdots)]
                                       + u1[index2D((i + 1), j, xdots)] + dgx * u1[index2D((i + 1), j, xdots)])
                                       + CY * (u1[index2D(i, (j - 1), xdots)]
                                       + u1[index2D(i, (j + 1), xdots)] + dgy * u1[index2D(i, j, xdots)]);
        }
    }

    for (j = 0; j < ydots - 1; j++) 
    {
        u2[index2D(0, j, xdots)] = u2[index2D(1, j, xdots)];
        u2[index2D(Xdots - 1, j, xdots)] = u2[index2D(Xdots - 2, j, xdots)];
    }

    for (i = 0; i < xdots - 1; i++) 
    {
        u2[index2D(i, 0, xdots)] = u2[index2D(i, 1, xdots)];
        u2[index2D(i, Ydots - 1, xdots)] = u2[index2D(i, Ydots - 2, xdots)];
    }

    return;
}